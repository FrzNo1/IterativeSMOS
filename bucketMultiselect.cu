#include "hip/hip_runtime.h"
/* Copyright 2012 Jeffrey Blanchard, Erik Opavsky, and Emircan Uysaler
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

namespace BucketMultiselect{
  using namespace std;

#define MAX_THREADS_PER_BLOCK 1024
#define CUTOFF_POINT 200000 

#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)


  /// ***********************************************************
  /// ***********************************************************
  /// **** HELPER CPU FUNCTIONS
  /// ***********************************************************
  /// ***********************************************************

  hipEvent_t start, stop;
  float time;

  /* start a timer with selection = 0
   * stop a timer with selection = 1
   */
  void timing(int selection, int ind){
    if(selection == 0) {
      //****//
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start, 0);
      //****//
    }
    else {
      //****//
      hipDeviceSynchronize();
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);
      hipEventDestroy(start);
      hipEventDestroy(stop);
      printf("Time %d: %lf \n", ind, time);
      //****//
    }
  }

  //This function initializes a vector to all zeros on the host (CPU)
  template<typename T>
  void setToAllZero(T * deviceVector, int length) {
    hipMemset(deviceVector, 0, length * sizeof(T));
  }


  /// ***********************************************************
  /// ***********************************************************
  /// **** HELPER GPU FUNCTIONS-KERNELS
  /// ***********************************************************
  /// ***********************************************************

  //this function assigns elements to buckets based off of a randomized sampling of the elements in the vector
  template <typename T>
  __global__ void assignSmartBucket(T * d_vector, int length, int numBuckets, double * slopes, T * pivots, int numPivots, uint* elementToBucket, uint* bucketCount, int offset){
  
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    uint bucketIndex;
    int threadIndex = threadIdx.x;  
    
    //variables in shared memory for fast access
    __shared__ int sharedNumSmallBuckets;
    if (threadIndex < 1) 
      sharedNumSmallBuckets = numBuckets / (numPivots-1);
    
    extern __shared__ uint array[];
    uint * sharedBuckets = (uint *)array;
    double * sharedSlopes = (double *)&sharedBuckets[numBuckets];
    T * sharedPivots = (T *)&sharedSlopes[numPivots-1];
    /*
    uint * sharedBuckets = (uint *)array;
    double * sharedSlopes = (double *)&sharedBuckets[numBuckets];
    T * sharedPivots = (T *)&sharedSlopes[numPivots-1];
    // statically allocating the array gives faster results
    __shared__ double sharedSlopes[NUM_PIVOTS-1];
    __shared__ T sharedPivots[NUM_PIVOTS];
    */
  
    //reading bucket counts into shared memory where increments will be performed
    for (int i = 0; i < (numBuckets / MAX_THREADS_PER_BLOCK); i++) 
      if (threadIndex < numBuckets) 
        sharedBuckets[i * MAX_THREADS_PER_BLOCK + threadIndex] = 0;

    if(threadIndex < numPivots) {
      *(sharedPivots + threadIndex) = *(pivots + threadIndex);
      if(threadIndex < numPivots-1) 
        sharedSlopes[threadIndex] = slopes[threadIndex];
    }
    syncthreads();

    //assigning elements to buckets and incrementing the bucket counts
    if(index < length) {
      int i;

      for(i = index; i < length; i += offset) {
        T num = d_vector[i];
        int minPivotIndex = 0;
        int maxPivotIndex = numPivots-1;
        int midPivotIndex;

        // find the index of the pivot that is the greatest s.t. lower than or equal to num using binary search
        //while (maxPivotIndex > minPivotIndex+1) {
        for(int j = 1; j < numPivots - 1; j*=2) {
          midPivotIndex = (maxPivotIndex + minPivotIndex) / 2;
          if (num >= sharedPivots[midPivotIndex])
            minPivotIndex = midPivotIndex;
          else
            maxPivotIndex = midPivotIndex;
        }

        /*
          minPivotIndex = (((num>=sharedPivots[0]) & (num<sharedPivots[1])) * 0) |
          (((num>=sharedPivots[1]) & (num<sharedPivots[2])) * 1) | 
          (((num>=sharedPivots[2]) & (num<sharedPivots[3])) * 2) | 
          (((num>=sharedPivots[3]) & (num<sharedPivots[4])) * 3) | 
          (((num>=sharedPivots[4]) & (num<sharedPivots[5])) * 4) | 
          (((num>=sharedPivots[5]) & (num<sharedPivots[6])) * 5) | 
          (((num>=sharedPivots[6]) & (num<sharedPivots[7])) * 6) | 
          (((num>=sharedPivots[7]) & (num<sharedPivots[8])) * 7) | 
          (((num>=sharedPivots[8]) & (num<sharedPivots[9])) * 8) | 
          (((num>=sharedPivots[9]) & (num<sharedPivots[10])) * 9) | 
          (((num>=sharedPivots[10]) & (num<sharedPivots[11])) * 10) | 
          (((num>=sharedPivots[11]) & (num<sharedPivots[12])) * 11) | 
          (((num>=sharedPivots[12]) & (num<sharedPivots[13])) * 12) | 
          (((num>=sharedPivots[13]) & (num<sharedPivots[14])) * 13) | 
          (((num>=sharedPivots[14]) & (num<sharedPivots[15])) * 14) | 
          (((num>=sharedPivots[15]) & (num<sharedPivots[16])) * 15) | 
          ((num>=sharedPivots[16]) * 16);
        */

        bucketIndex = (minPivotIndex * sharedNumSmallBuckets) + (int) ((num - sharedPivots[minPivotIndex]) * sharedSlopes[minPivotIndex]);
        elementToBucket[i] = bucketIndex;
        // hashmap implementation set[bucketindex]=add.i;
        //bucketCount[blockIdx.x * numBuckets + bucketIndex]++;
        atomicInc (sharedBuckets + bucketIndex, length);
      }
    }
    
    syncthreads();

    //reading bucket counts from shared memory back to global memory
    for (int i = 0; i < (numBuckets / MAX_THREADS_PER_BLOCK); i++) 
      if (threadIndex < numBuckets) 
        //atomicAdd(bucketCount + blockIdx.x * numBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex, sharedBuckets[i * MAX_THREADS_PER_BLOCK + threadIndex]);
        *(bucketCount + blockIdx.x * numBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex) = *(sharedBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex);
  }
 

  //this function finds the bin containing the kth element we are looking for (works on the host)
  inline int findKBuckets(uint * d_bucketCount, uint * h_bucketCount, int numBuckets, uint * kVals, int kCount, uint * sums, uint * kthBuckets, int numBlocks){
    int sumsRowIndex= numBuckets * (numBlocks-1);
    /*
      for(int j=0; j<numBuckets; j++)
      CUDA_CALL(hipMemcpy(h_bucketCount + j, d_bucketCount + sumsRowIndex + j, sizeof(uint), hipMemcpyDeviceToHost));
    */
    CUDA_CALL(hipMemcpy(h_bucketCount, d_bucketCount + sumsRowIndex, sizeof(uint) * numBuckets, hipMemcpyDeviceToHost));

    int kBucket = 0;
    int k;
    int sum = h_bucketCount[0];

    for(register int i = 0; i < kCount; i++) {
      k = kVals[i];
      while ((sum < k) & (kBucket < numBuckets - 1)) {
        kBucket++;
        sum += h_bucketCount[kBucket];
      }
      kthBuckets[i] = kBucket;
      sums[i] = sum - h_bucketCount[kBucket];
    }

    return 0;
  }

  __global__ void sumCounts(uint * d_bucketCount, const int numBuckets, const int numBlocks) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j=1; j<numBlocks; j++) 
      d_bucketCount[index + numBuckets*j] += d_bucketCount[index + numBuckets*(j-1)];
    
  }

  __global__ void reindexCounts(uint * d_bucketCount, const int numBuckets, const int numBlocks, uint * d_reindexCounter, uint * d_markedBuckets, const int numUniqueBuckets) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex<numUniqueBuckets) {
      int index = d_markedBuckets[threadIndex];
      int add = d_reindexCounter[threadIndex];

      for(int j=0; j<numBlocks; j++) 
        d_bucketCount[index + numBuckets*j] += (uint) add;
    }
  }


  //copy elements in the kth bucket to a new array
  template <typename T>
  __global__ void copyElements (T* d_vector, int length, uint* elementToBucket, uint * buckets, const int numBuckets, T* newArray, uint offset, uint * d_bucketCount, int numTotalBuckets){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex;
    int loop = numBuckets / MAX_THREADS_PER_BLOCK;

    extern __shared__ uint array[];
    uint * sharedBucketCounts= (uint*)array;
    uint * sharedBuckets= (uint*)&array[numBuckets];

    for (int i = 0; i <= loop; i++) {      
      threadIndex = i * blockDim.x + threadIdx.x;
      if(threadIndex < numBuckets) {
        sharedBuckets[threadIndex]=buckets[threadIndex];
        sharedBucketCounts[threadIndex] = d_bucketCount[blockIdx.x * numTotalBuckets + sharedBuckets[threadIndex]];
      }
    }
    
    syncthreads();

    int minBucketIndex;
    int maxBucketIndex; 
    int midBucketIndex;
    uint temp;
    //uint holder;

    if(idx < length) {
      for(int i=idx; i<length; i+=offset) {
        temp = elementToBucket[i];
        minBucketIndex = 0;
        maxBucketIndex = numBuckets-1;

        //copy elements in the kth buckets to the new array
        for(int j = 1; j < numBuckets; j*=2) {  
          midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
          if (temp > sharedBuckets[midBucketIndex])
            minBucketIndex=midBucketIndex+1;
          else
            maxBucketIndex=midBucketIndex;
        }

        if (buckets[maxBucketIndex] == temp) 
          //newArray[atomicDec(d_bucketCount + blockIdx.x * numTotalBuckets + temp, length)-1] = d_vector[i];
          newArray[atomicDec(sharedBucketCounts + maxBucketIndex, length)-1] = d_vector[i];
      }
    }

  }

  /// ***********************************************************
  /// ***********************************************************
  /// **** GENERATE PIVOTS
  /// ***********************************************************
  /// ***********************************************************

  __host__ __device__
  unsigned int hash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
  }

  struct RandomNumberFunctor :
    public thrust::unary_function<unsigned int, float> {
    unsigned int mainSeed;

    RandomNumberFunctor(unsigned int _mainSeed) :
      mainSeed(_mainSeed) {}
  
    __host__ __device__
    float operator()(unsigned int threadIdx)
    {
      unsigned int seed = hash(threadIdx) * mainSeed;

      thrust::default_random_engine rng(seed);
      rng.discard(threadIdx);
      thrust::uniform_real_distribution<float> u(0, 1);

      return u(rng);
    }
  };

  template <typename T>
  void createRandomVector(T * d_vec, int size) {
    timeval t1;
    uint seed;

    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
  
    thrust::device_ptr<T> d_ptr(d_vec);
    thrust::transform (thrust::counting_iterator<uint>(0),thrust::counting_iterator<uint>(size), d_ptr, RandomNumberFunctor(seed));
  }

  template <typename T>
  __global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
    *(in + blockIdx.x*blockDim.x + threadIdx.x) = *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
  }


  __global__ void enlargeIndexAndGetElements (float * in, uint * out, uint * list, int size) {
    *(out + blockIdx.x * blockDim.x + threadIdx.x) = (uint) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
  }

  template <typename T>
  void generatePivots (uint * pivots, double * slopes, uint * d_list, int sizeOfVector, int numPivots, int sizeOfSample, int totalSmallBuckets, uint min, uint max) {
  
    float * d_randomFloats;
    uint * d_randomInts;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);
  
    d_randomInts = (uint *) d_randomFloats;

    createRandomVector (d_randomFloats, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK), MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list, sizeOfVector);

    pivots[0] = min;
    pivots[numPivots-1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randomInts);
    thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are next to the min and max pivots using the random element endOffset away from the ends
    hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (uint), hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1, sizeof (uint), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

    for (register int i = 2; i < numPivots - 2; i++) {
      hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1, sizeof (uint), hipMemcpyDeviceToHost);
      slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
    }

    slopes[numPivots - 3] = numSmallBuckets / (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets / (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

    hipFree(d_randomFloats);
    hipFree(d_randomInts);
  }
  
  template <typename T>
  void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector, int numPivots, int sizeOfSample, int totalSmallBuckets, T min, T max) {
    T * d_randoms;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randoms, sizeof (T) * sizeOfSample);
  
    createRandomVector (d_randoms, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK), MAX_THREADS_PER_BLOCK>>>(d_randoms, d_list, sizeOfVector);

    pivots[0] = min;
    pivots[numPivots - 1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randoms);
    thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are endOffset away from the min and max pivots
    hipMemcpy (pivots + 1, d_randoms + endOffset - 1, sizeof (T), hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randoms + sizeOfSample - endOffset - 1, sizeof (T), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

    for (register int i = 2; i < numPivots - 2; i++) {
      hipMemcpy (pivots + i, d_randoms + pivotOffset * (i - 1) + endOffset - 1, sizeof (T), hipMemcpyDeviceToHost);
      slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
    }

    slopes[numPivots - 3] = numSmallBuckets / (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets / (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);
  
    hipFree(d_randoms);
  }

  // copyValuesInChunk<T>(output, d_output, newInput, d_kList, d_kIndices, kListCount);
  // for (register int i = 0; i < kListCount; i++) 
  //   CUDA_CALL(hipMemcpy(output + kIndices[i], newInput + kList[i] - 1, sizeof (T), hipMemcpyDeviceToHost));
  template <typename T>
  __global__ void copyValuesInChunk (T * outputVector, T * inputVector, uint * kList, uint * kIndices, int kListCount) {
   
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < kListCount)
      *(outputVector + *(kIndices + idx)) = *(inputVector + *(kList + idx) - 1);

    /*
    if (idx == 0)
      printf ("idx = 0\n");
    if (idx == kListCount - 1)
    printf ("idx = %d\n", kListCount - 1);*/
    // if (*(kIndices + idx) == kListCount - 1)
    //  printf ("output[%d] = %f\n", *(kIndices + idx), *(outputVector + *(kIndices + idx)));
  }



  /// ***********************************************************
  /// ***********************************************************
  /// **** PHASE ONE
  /// ***********************************************************
  /// ***********************************************************

  /* this function finds the kth-largest element from the input array */
  template <typename T>
  T phaseOne (T* d_vector, int length, uint * kList, int kListCount, T * output, int blocks, int threads, int numBuckets, int numPivots) {    
    /// ***********************************************************
    /// ****STEP 1: Find Min and Max of the whole vector
    /// ****We don't need to go through the rest of the algorithm if it's flat
    /// ***********************************************************
    // timing(0, 1);

    //find max and min with thrust
    T maximum, minimum;

    thrust::device_ptr<T>dev_ptr(d_vector);
    thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result = thrust::minmax_element(dev_ptr, dev_ptr + length);

    minimum = *result.first;
    maximum = *result.second;

    //if the max and the min are the same, then we are done
    if (maximum == minimum) {
      for (register int i = 0; i < kListCount; i++) 
        output[i] = minimum;
      
      return 1;
    }

    // timing(1, 1);
    /// ***********************************************************
    /// ****STEP 2: Declare variables and allocate memory
    /// **** Declare Variables
    /// ***********************************************************
    // timing(0, 2);

    //declaring variables for kernel launches
    int threadsPerBlock = threads;
    int numBlocks = blocks;
    int offset = blocks * threads;

    // variables for the randomized selection
    int sampleSize = 1024;

    // pivot variables
    double slopes[numPivots - 1];
    double * d_slopes;
    T pivots[numPivots];
    T * d_pivots;

    //Allocate memory to store bucket assignments
    size_t size = length * sizeof(uint);
    uint * d_elementToBucket;    //array showing what bucket every element is in

    CUDA_CALL(hipMalloc(&d_elementToBucket, size));

    //Allocate memory to store bucket counts
    size_t totalBucketSize = numBlocks * numBuckets * sizeof(uint);
    uint h_bucketCount[numBuckets]; //array showing the number of elements in each bucket
    uint * d_bucketCount; 

    CUDA_CALL(hipMalloc(&d_bucketCount, totalBucketSize));
    //setToAllZero<uint>(d_bucketCount, numBlocks * numBuckets);

    // array of kth buckets
    int numUniqueBuckets;
    uint * d_kList; 
    uint kthBuckets[kListCount]; 
    uint kthBucketScanner[kListCount]; 
    uint * kIndices = (uint *) malloc (kListCount * sizeof (uint));
    uint * d_kIndices;
    uint uniqueBuckets[kListCount];
    uint * d_uniqueBuckets; 
    uint reindexCounter[kListCount];  
    uint * d_reindexCounter;    

    CUDA_CALL(hipMalloc(&d_kList, kListCount * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_kIndices, kListCount * sizeof (uint)));

    for (register int i = 0; i < kListCount; i++) {
      kthBucketScanner[i] = 0;
      kIndices[i] = i;
    }

    // variable to store the end result
    int newInputLength;
    T* newInput;

    // timing(1, 2);
    /// ***********************************************************
    /// ****STEP 3: Sort the klist
    /// and keep the old index
    /// ***********************************************************
    // timing(0, 3);

    CUDA_CALL(hipMemcpy(d_kIndices, kIndices, kListCount * sizeof (uint), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_kList, kList, kListCount * sizeof (uint), hipMemcpyHostToDevice)); 

    // sort the given indices
    thrust::device_ptr<uint>kList_ptr(d_kList);
    thrust::device_ptr<uint>kIndices_ptr(d_kIndices);
    thrust::sort_by_key(kList_ptr, kList_ptr + kListCount, kIndices_ptr);

    CUDA_CALL(hipMemcpy(kIndices, d_kIndices, kListCount * sizeof (uint), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(kList, d_kList, kListCount * sizeof (uint), hipMemcpyDeviceToHost)); 

    //  hipFree(d_kIndices); 
    //  hipFree(d_kList); 
    
    int kOffset = kListCount - 1;
    while (kList[kOffset] == length) {
      output[kIndices[kListCount-1]] = maximum;
      kListCount--;
      kOffset--;
    }

    kOffset = 0;
    while (kList[0] == 1) {
      output[kIndices[0]] = minimum;
      kIndices++;
      kList++;
      kListCount--;
      kOffset++;
    }

    for (int x = 0; x < kListCount + kOffset; x++)
      printf ("output[%d] = %f\n", x, output[kIndices[x] - 1]);

    // timing(1, 3);
    /// ***********************************************************
    /// ****STEP 4: Generate Pivots and Slopes
    /// Declare slopes and pivots
    /// ***********************************************************
    // timing(0, 4);

    CUDA_CALL(hipMalloc(&d_slopes, (numPivots - 1) * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_pivots, numPivots * sizeof(T)));

    // Find bucket sizes using a randomized selection
    generatePivots<T>(pivots, slopes, d_vector, length, numPivots, sampleSize, numBuckets, minimum, maximum);
    
    // make any slopes that were infinity due to division by zero (due to no 
    //  difference between the two associated pivots) into zero, so all the
    //  values which use that slope are projected into a single bucket
    for (register int i = 0; i < numPivots - 1; i++)
      if (isinf(slopes[i]))
        slopes[i] = 0;

    CUDA_CALL(hipMemcpy(d_slopes, slopes, (numPivots - 1) * sizeof(double), hipMemcpyHostToDevice));  
    CUDA_CALL(hipMemcpy(d_pivots, pivots, numPivots * sizeof(T), hipMemcpyHostToDevice));

    // timing(1, 4);
    /// ***********************************************************
    /// ****STEP 5: Assign elements to buckets
    /// 
    /// ***********************************************************
    // timing(0, 5);

    //Distribute elements into their respective buckets
    assignSmartBucket<T><<<numBlocks, threadsPerBlock,  numPivots * sizeof(T) + (numPivots-1) * sizeof(double) + numBuckets * sizeof(uint)>>>(d_vector, length, numBuckets, d_slopes, d_pivots, numPivots, d_elementToBucket, d_bucketCount, offset);
    // timing(1, 5);
    // timing(0, 21);

    sumCounts<<<numBuckets/threadsPerBlock, threadsPerBlock>>>(d_bucketCount, numBuckets, numBlocks);

    // timing(1, 21);
    /// ***********************************************************
    /// ****STEP 6: Find the kth buckets
    /// and their respective update indices
    /// ***********************************************************
    // timing(0, 6);

    findKBuckets(d_bucketCount, h_bucketCount, numBuckets, kList, kListCount, kthBucketScanner, kthBuckets, numBlocks);

    // timing(1, 6);
    // timing(0, 7);

    // we must update K since we have reduced the problem size to elements in the kth bucket
    //  get the index of the first element
    //  add the number of elements
    uniqueBuckets[0] = kthBuckets[0];
    reindexCounter[0] = 0;
    numUniqueBuckets = 1;
    kList[0] -= kthBucketScanner[0];

    for (int i = 1; i < kListCount; i++) {
      if (kthBuckets[i] != kthBuckets[i-1]) {
        uniqueBuckets[numUniqueBuckets] = kthBuckets[i];
        reindexCounter[numUniqueBuckets] = reindexCounter[numUniqueBuckets-1] + h_bucketCount[kthBuckets[i-1]];
        numUniqueBuckets++;
      }
      kList[i] = reindexCounter[numUniqueBuckets-1] + kList[i] - kthBucketScanner[i];
    }

    newInputLength = reindexCounter[numUniqueBuckets-1] + h_bucketCount[kthBuckets[kListCount - 1]];

    // printf("bucketmultiselectBlocked total kbucket_count = %d\n", newInputLength);
    // printf("numMarkedBuckets = %d\n", numUniqueBuckets);

    // timing(1, 7);
    // timing(0, 22);

    CUDA_CALL(hipMalloc(&d_reindexCounter, numUniqueBuckets * sizeof(uint)));
    CUDA_CALL(hipMalloc(&d_uniqueBuckets, numUniqueBuckets * sizeof(uint)));

    CUDA_CALL(hipMemcpy(d_reindexCounter, reindexCounter, numUniqueBuckets * sizeof(uint), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_uniqueBuckets, uniqueBuckets, numUniqueBuckets * sizeof(uint), hipMemcpyHostToDevice));

    reindexCounts<<<ceil((float)numUniqueBuckets/threadsPerBlock), threadsPerBlock>>>(d_bucketCount, numBuckets, numBlocks, d_reindexCounter, d_uniqueBuckets, numUniqueBuckets);

    // timing(1, 22);
    /// ***********************************************************
    /// ****STEP 7: Copy the kth buckets
    /// only unique ones
    /// ***********************************************************
    // timing(0, 8);

    // allocate memories
    CUDA_CALL(hipMalloc(&newInput, newInputLength * sizeof(T)));
   
    // timing(1, 8);
    // timing(0, 9);
 
    //copyElements<<<numBlocks, threadsPerBlock, numUniqueBuckets * sizeof(uint)>>>(d_vector, length, d_elementToBucket, d_uniqueBuckets, numUniqueBuckets, newInput, d_uniqueBucketIndexCounter, offset);
    copyElements<T><<<numBlocks, threadsPerBlock, numUniqueBuckets * 2 * sizeof(uint)>>>(d_vector, length, d_elementToBucket, d_uniqueBuckets, numUniqueBuckets, newInput, offset, d_bucketCount, numBuckets);
  
    // timing(1, 9);

    /// ***********************************************************
    /// ****STEP 8: Sort
    /// and finito
    /// ***********************************************************

    //free all used memory
    hipFree(d_pivots);
    hipFree(d_slopes); 

    hipFree(d_elementToBucket);  
    hipFree(d_bucketCount); 
    hipFree(d_uniqueBuckets); 
    hipFree(d_reindexCounter);  

    // timing(0, 10);
    // sort the vector
    thrust::device_ptr<T>newInput_ptr(newInput);
    thrust::sort(newInput_ptr, newInput_ptr + newInputLength);


    // printf("newInputLength = %d\n", newInputLength);
    // for (register int i = 0; i < kListCount; i++) 
    //  CUDA_CALL(hipMemcpy(output + kIndices[i], newInput + kList[i] - 1, sizeof (T), hipMemcpyDeviceToHost));

    /* new strategy for copying k values back in a chunk */
    
  

    T * d_output;
    CUDA_CALL(hipMalloc (&d_output, (kListCount + kOffset) * sizeof (T)));
    CUDA_CALL(hipMemcpy (d_output, output, (kListCount + kOffset) * sizeof (T), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy (d_kList, kList, (kListCount + kOffset) * sizeof (uint), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy (d_kIndices, kIndices, (kListCount + kOffset) * sizeof (uint), hipMemcpyHostToDevice));

   

    threads = MAX_THREADS_PER_BLOCK;
    if (kListCount < threads)
      threads = kListCount;
    blocks = (int) ceil (kListCount / (float) threads);

    copyValuesInChunk<<<blocks, threads>>>(d_output, newInput, d_kList, d_kIndices, kListCount);

    hipMemcpy (output, d_output, kListCount * sizeof (T), hipMemcpyDeviceToHost);

    hipFree(d_output);
    hipFree(d_kIndices); 
    hipFree(d_kList); 
    

    /* done new strategy for copying k values back in a chunk */

    // timing(1, 10);

    hipFree(newInput); 
    free (kIndices - kOffset);

    return 1;
  }

  template <typename T>
  T bucketMultiselectWrapper (T * d_vector, int length, uint * kList_ori, int kListCount, T * outputs, int blocks, int threads) { 

    int numBuckets = 8192;
    uint kList[kListCount];

    // turn it into kth smallest
    for (register int i = 0; i < kListCount; i++) 
      kList[i] = length - kList_ori[i] + 1;
   
    // optimize numBuckets based on experimentation
    if (length <= 524288)
      numBuckets = 4096;
    else if (length <= 1048576 && kListCount <= 188)
      numBuckets = 4096;
    else if (length <= 2097152 && kListCount <= 94)
      numBuckets = 4096;
    else if (length <= 4194304 && kListCount <= 48)
      numBuckets = 4096;
    else if (length <= 8388608 && kListCount <= 20)
      numBuckets = 4096;
    else if (length <= 16777216 && kListCount <= 10)
      numBuckets = 4096;
    else if (length <= 33554432 && kListCount <= 6)
      numBuckets = 4096;
    else if (length <= 67108864 && kListCount <= 4)
      numBuckets = 4096;

   
    phaseOne (d_vector, length, kList, kListCount, outputs, blocks, threads, numBuckets, 17);

    return 1;
  }
}

