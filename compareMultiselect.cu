/* Based on compareAlgorithms.cu */

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>

#include <algorithm>
//Include various thrust items that are used
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/random.h>

//various functions, include the functions
//that print numbers in binary.
#include "printFunctions.cu"

//the algorithms
#include "bucketMultiselect.cu"
#include "naiveBucketMultiselect.cu"

#include "generateProblems.cu"
#include "multiselectTimingFunctions.cu"

#define NUMBEROFALGORITHMS 3
char* namesOfMultiselectTimingFunctions[NUMBEROFALGORITHMS] = {"Sort and Choose Multiselect", "Bucket Multiselect", "Naive Bucket Multiselect"};


using namespace std;
template<typename T>
void compareMultiselectAlgorithms(uint size, uint * kVals, uint kListCount, uint numTests, uint *algorithmsToTest, uint generateType, uint kGenerateType, char* fileNamecsv) {
  T *h_vec, *h_vec_copy;
  float timeArray[NUMBEROFALGORITHMS][numTests];
  T * resultsArray[NUMBEROFALGORITHMS][numTests];
  float totalTimesPerAlgorithm[NUMBEROFALGORITHMS];
  uint winnerArray[numTests];
  uint timesWon[NUMBEROFALGORITHMS];
  uint i,j,m,x;
  int runOrder[NUMBEROFALGORITHMS];

  unsigned long long seed;
  results_t<T> *temp;
  ofstream fileCsv;
  timeval t1;
 
  typedef results_t<T>* (*ptrToTimingFunction)(T*, uint, uint *, uint);
  typedef void (*ptrToGeneratingFunction)(T*, uint, hiprandGenerator_t);

  //these are the functions that can be called
  ptrToTimingFunction arrayOfTimingFunctions[NUMBEROFALGORITHMS] = {&timeSortAndChooseMultiselect<T>,
                                                                    &timeBucketMultiselect<T>, 
                                                                    &timeNaiveBucketMultiselect<T>};
  
  ptrToGeneratingFunction *arrayOfGenerators;
  char** namesOfGeneratingFunctions;
  //this is the array of names of functions that generate problems of this type, ie float, double, or uint
  namesOfGeneratingFunctions = returnNamesOfGenerators<T>();
  arrayOfGenerators = (ptrToGeneratingFunction *) returnGenFunctions<T>();

  printf("Files will be written to %s\n", fileNamecsv);
  fileCsv.open(fileNamecsv, ios_base::app);
  
  //zero out the totals and times won
  bzero(totalTimesPerAlgorithm, NUMBEROFALGORITHMS * sizeof(uint));
  bzero(timesWon, NUMBEROFALGORITHMS * sizeof(uint));

  //allocate space for h_vec, and h_vec_copy
  h_vec = (T *) malloc(size * sizeof(T));
  h_vec_copy = (T *) malloc(size * sizeof(T));

  //create the random generator.
  hiprandGenerator_t generator;
  srand(unsigned(time(NULL)));

  printf("The distribution is: %s\n", namesOfGeneratingFunctions[generateType]);
  printf("The k distribution is: %s\n", namesOfKGenerators[kGenerateType]);

  //*/******************* START RUNNING TESTS *************
  /***********************************************/

  for(i = 0; i < numTests; i++) {
    // hipDeviceReset();
    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
    
    for(m = 0; m < NUMBEROFALGORITHMS;m++)
      runOrder[m] = m;
    
    std::random_shuffle(runOrder, runOrder + NUMBEROFALGORITHMS);
    //fileCsv << size << "," << kVals[0] << "," << kVals[kListCount - 1] << "," << kListCount << "," << (100*((float)kListCount/size)) << "," << namesOfGeneratingFunctions[generateType] << "," << namesOfKGenerators[kGenerateType] << "," << seed << ",";
    fileCsv << size << "," << kListCount << "," << namesOfGeneratingFunctions[generateType] << "," << namesOfKGenerators[kGenerateType] << ",";
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator,seed);
    printf("Running test %u of %u for size: %u and numK: %u\n", i + 1, numTests, size, kListCount);
    //generate the random vector using the specified distribution
    arrayOfGenerators[generateType](h_vec, size, generator);

    //copy the vector to h_vec_copy, which will be used to restore it later
    memcpy(h_vec_copy, h_vec, size * sizeof(T));

    winnerArray[i] = 0;
    float currentWinningTime = INFINITY;
    //run the various timing functions
    for(x = 0; x < NUMBEROFALGORITHMS; x++){
      j = runOrder[x];
      if(algorithmsToTest[j]){

        //run timing function j
        printf("TESTING: %u\n", j);
        temp = arrayOfTimingFunctions[j](h_vec_copy, size, kVals, kListCount);

        //record the time result
        timeArray[j][i] = temp->time;
        //record the value returned
        resultsArray[j][i] = temp->vals;
        //update the current "winner" if necessary
        if(timeArray[j][i] < currentWinningTime){
          currentWinningTime = temp->time;
          winnerArray[i] = j;
        }

        //perform clean up 
        free(temp);
        memcpy(h_vec_copy, h_vec, size * sizeof(T));
      }
    }

    hiprandDestroyGenerator(generator);
    for(x = 0; x < NUMBEROFALGORITHMS; x++)
      if(algorithmsToTest[x])
        fileCsv << namesOfMultiselectTimingFunctions[x] << "," << timeArray[x][i] << ",";

    uint flag = 0;
    for(m = 1; m < NUMBEROFALGORITHMS;m++)
      if(algorithmsToTest[m])
        for (j = 0; j < kListCount; j++) {
          if(resultsArray[m][i][j] != resultsArray[0][i][j]) {
            flag++;
            fileCsv << "\nERROR ON TEST " << i << " of " << numTests << " tests!!!!!\n";
            fileCsv << "vector size = " << size << "\nvector seed = " << seed << "\n";
            fileCsv << "kListCount = " << kListCount << "\n";
            fileCsv << "wrong k = " << kVals[j] << " kIndex = " << j << " wrong result = " << resultsArray[m][i][j] << " correct result = " <<  resultsArray[0][i][j] << "\n";
            std::cout <<namesOfMultiselectTimingFunctions[m] <<" did not return the correct answer on test " << i + 1 << " at k[" << j << "].  It got "<< resultsArray[m][i][j];
            std::cout << " instead of " << resultsArray[0][i][j] << ".\n" ;
            std::cout << "RESULT:\t";
            PrintFunctions::printBinary(resultsArray[m][i][j]);
            std::cout << "Right:\t";
            PrintFunctions::printBinary(resultsArray[0][i][j]);
          }
        }

    fileCsv << flag << "\n";
  }
  
  //calculate the total time each algorithm took
  for(i = 0; i < numTests; i++)
    for(j = 0; j < NUMBEROFALGORITHMS;j++)
      if(algorithmsToTest[j])
        totalTimesPerAlgorithm[j] += timeArray[j][i];


  //count the number of times each algorithm won. 
  for(i = 0; i < numTests;i++)
    timesWon[winnerArray[i]]++;

  printf("\n\n");

  //print out the average times
  for(i = 0; i < NUMBEROFALGORITHMS; i++)
    if(algorithmsToTest[i])
      printf("%-20s averaged: %f ms\n", namesOfMultiselectTimingFunctions[i], totalTimesPerAlgorithm[i] / numTests);

  for(i = 0; i < NUMBEROFALGORITHMS; i++)
    if(algorithmsToTest[i])
      printf("%s won %u times\n", namesOfMultiselectTimingFunctions[i], timesWon[i]);


  // free results
  for(i = 0; i < numTests; i++) 
    for(m = 0; m < NUMBEROFALGORITHMS; m++) 
      if(algorithmsToTest[m])
        free(resultsArray[m][i]);

  //free h_vec and h_vec_copy
  free(h_vec);
  free(h_vec_copy);
  //close the file
  fileCsv.close();
}


template<typename T>
void runTests (uint generateType, char* fileName, uint startPower, uint stopPower, uint timesToTestEachK, 
               uint kDistribution, uint startK, uint stopK, uint kJump) {
  uint algorithmsToRun[NUMBEROFALGORITHMS]= {1, 1, 0};
  uint size;
  uint i;
  uint arrayOfKs[stopK+1];
  
  
  for(size = (1 << startPower); size <= (1 << stopPower); size *= 2) {
    /*
    //calculate k values
    arrayOfKs[0] = 2;
    //  arrayOfKs[1] = (uint) (.01 * (float) size);
    //  arrayOfKs[2] = (uint) (.025 * (float) size);
    for(i = 1; i <= num - 2; i++) 
    arrayOfKs[i] = (uint) (( i / (float) num ) * size);
    
    //  arrayOfKs[num-3] = (uint) (.9975 * (float) size);
    //  arrayOfKs[num-2] = (uint) (.999 * (float) size);
    arrayOfKs[num-1] = (uint) (size - 2); 
    */
    unsigned long long seed;
    timeval t1;
    gettimeofday(&t1, NULL);
    seed = t1.tv_usec * t1.tv_sec;
    hiprandGenerator_t generator;
    srand(unsigned(time(NULL)));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator,seed);

    arrayOfKDistributionGenerators[kDistribution](arrayOfKs, stopK, size, generator);

    hiprandDestroyGenerator(generator);

    /*
    printf("arrayOfKs = ");
    for(uint j = 0; j < stopK+1; j++)
      printf("%u; ", arrayOfKs[j]);
    printf("\n\n");
    */

    for(i = startK; i <= stopK; i+=kJump) {
      hipDeviceReset();
      hipDeviceReset();
      printf("NOW ADDING ANOTHER K\n\n");
      compareMultiselectAlgorithms<T>(size, arrayOfKs, i, timesToTestEachK, algorithmsToRun, generateType, kDistribution, fileName);
    }
  }
}


int main (int argc, char *argv[]) {
  char *fileName, *hostName, *typeString;

  uint testCount;
  fileName = (char*) malloc(128 * sizeof(char));
  typeString = (char*) malloc(10 * sizeof(char));
  hostName = (char*) malloc(20 * sizeof(char));
  gethostname(hostName, 20);

  time_t rawtime;
  struct tm * timeinfo;
  time ( &rawtime );
  timeinfo = localtime ( &rawtime );
  char * humanTime = asctime(timeinfo);
  humanTime[strlen(humanTime)-1] = '\0';

  uint type,distributionType,startPower,stopPower,kDistribution,startK,stopK,jumpK;
  
  printf("Please enter the type of value you want to test:\n0-float\n1-double\n2-uint\n");
  scanf("%u", &type);
  printf("Please enter distribution type: ");
  printDistributionOptions(type);
  scanf("%u", &distributionType);
  printf("Please enter K distribution type: ");
  printKDistributionOptions();
  scanf("%u", &kDistribution);
  printf("Please enter Start power: ");
  scanf("%u", &startPower);
  printf("Please enter Stop power: ");
  scanf("%u", &stopPower); 
  printf("Please enter Start number of K values: ");
  scanf("%u", &startK);
  printf("Please enter number of K values to jump by: ");
  scanf("%u", &jumpK);
  printf("Please enter Stop number of K values: ");
  scanf("%u", &stopK);
  printf("Please enter number of tests to run per K: ");
  scanf("%u", &testCount);

  switch(type){
  case 0:
    typeString = "float";
    break;
  case 1:
    typeString = "double";
    break;
  case 2:
    typeString = "uint";
    break;
  default:
    break;
  }

  snprintf(fileName, 128, "%s %s k-dist:%s 2^%d to 2^%d (%d:%d:%d) %d-tests on %s at %s", typeString, getDistributionOptions(type, distributionType), getKDistributionOptions(kDistribution), startPower, stopPower, startK, jumpK, stopK, testCount, hostName, humanTime);
  printf("File Name: %s \n", fileName);
  //printf("Please enter filename now: ");

  switch(type){
  case 0:
    runTests<float>(distributionType,fileName,startPower,stopPower,testCount,kDistribution,startK,stopK,jumpK);
    break;
  case 1:
    runTests<double>(distributionType,fileName,startPower,stopPower,testCount,kDistribution,startK,stopK,jumpK);
    break;
  case 2:
    runTests<uint>(distributionType,fileName,startPower,stopPower,testCount,kDistribution,startK,stopK,jumpK);
    break;
  default:
    printf("You entered and invalid option, now exiting\n");
    break;
  }

  free (fileName);
  return 0;
}
