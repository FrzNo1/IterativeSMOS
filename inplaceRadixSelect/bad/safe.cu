#include "hip/hip_runtime.h"
#include <thrust/scan.h>
  #include <thrust/count.h>
  #include <thrust/reduce.h>

namespace DestructiveRadixSelect{

  struct problemInfo_t{
    uint blocks;
    uint threadsPerBlock;
    uint totalThreads;
    uint currentSize;
    uint nextSize;
    uint k;
    uint countSize;
    uint numSmaller;
    uint previousDigit;
  };

template<typename T, uint RADIX_SIZE, uint BIT_SHIFT>
__global__ void getCounts(T *d_vec,const uint size, uint *digitCounts,const uint offset){ 
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int i;

  extern __shared__ ushort sharedCounts[];

  for(i = 0; i < 16;i++){
    sharedCounts[i * blockDim.x + threadIdx.x] = 0;
  }

  //We only look at the current digit, becasue it must be the case that 
  //all elements in d_vec share have the same digit for all digit places before the current one
  for(i = idx; i < size; i += offset){
    sharedCounts[(d_vec[i] >> BIT_SHIFT) * blockDim.x + threadIdx.x]++;
  }

  for(i = 0; i <16 ;i++){
    digitCounts[i * offset + idx] = sharedCounts[blockDim.x *i + threadIdx.x];
  }
  

}

template<typename T, uint RADIX_SIZE, uint BIT_SHIFT>
__global__ void getCounts2(T *d_vec,const uint size, uint *digitCounts,const uint offset, uint answerSoFar){ 
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int i;

  extern __shared__ ushort sharedCounts[];
  T value;
  for(i = 0; i < 16;i++){
    sharedCounts[i * blockDim.x + threadIdx.x] = 0;
  }

  //We only look at the current digit, becasue it must be the case that 
  //all elements in d_vec share have the same digit for all digit places before the current one
  for(i = idx; i < size; i += offset){
    value = d_vec[i];
    if((value >> (BIT_SHIFT + 4)) == (answerSoFar >> (BIT_SHIFT + 4))){
      sharedCounts[((d_vec[i] >> BIT_SHIFT) & 0xF) * blockDim.x + threadIdx.x]++;
    }
  }
  for(i = 0; i <16 ;i++){
    digitCounts[i * offset + idx] = sharedCounts[blockDim.x *i + threadIdx.x];
  }
  

}

template<typename T, uint RADIX_SIZE, uint BIT_SHIFT>
__global__ void getCountsWithShrink(T *d_vec,T * alt_vec,const uint size, uint *digitCounts,const uint offset,uint numSmaller,uint previousDigit, uint answerSoFar){ 
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int i;
  T value;
  int startIndex = digitCounts[ previousDigit * offset + idx] - numSmaller;
  extern __shared__ ushort sharedCounts[];

  for(i = 0; i < 16;i++){
    sharedCounts[i * blockDim.x + threadIdx.x] = 0;
  }

  for(i = idx; i < size; i += offset){
      value = d_vec[i];
      if( (value >> (BIT_SHIFT + 4))  == (answerSoFar >> (BIT_SHIFT + 4))){

       alt_vec[startIndex++] = value;
        sharedCounts[((value >> BIT_SHIFT) & 0xF) * blockDim.x + threadIdx.x]++;

      }
  }

  syncthreads();

  for(i = 0; i <16 ;i++){
    digitCounts[i * offset + idx] = sharedCounts[blockDim.x *i + threadIdx.x];
  }
  

}



  template<typename T,uint RADIX_SIZE, uint BIT_SHIFT, uint UPDATE>
uint determineDigit(uint *digitCounts, problemInfo_t &info){
  uint *numLessThanOrEqualTo;
  numLessThanOrEqualTo = (uint *) malloc(16 * sizeof(uint));
  uint *ns;
  ns = (uint *) malloc(sizeof(uint));
  uint i=0;
  if(UPDATE){
    info.currentSize = info.nextSize;
  }
  uint threshold = info.nextSize - info.k + 1;
  thrust::device_ptr<uint>ptr(digitCounts);
  thrust::exclusive_scan(ptr, ptr + (info.totalThreads * 16)+1, ptr);
  

  for(i =0; i < 16; i++){
    hipMemcpy(numLessThanOrEqualTo + i, digitCounts + ((i+1) * info.totalThreads), sizeof(uint), hipMemcpyDeviceToHost);
  }
 

 //identify the kth largest digit
  if(numLessThanOrEqualTo[0] >= threshold){
    info.k -= info.nextSize - numLessThanOrEqualTo[0];
    info.nextSize = numLessThanOrEqualTo[0];
    info.numSmaller = 0;
    return 0;

  }

  for(i = 1; i < 16; i++){
    if(numLessThanOrEqualTo[i] >= threshold){
      info.k -= info.nextSize - numLessThanOrEqualTo[i];
      info.nextSize = numLessThanOrEqualTo[i] - numLessThanOrEqualTo[i -1];
      hipMemcpy(ns, digitCounts + (i * info.totalThreads) , sizeof(uint), hipMemcpyDeviceToHost);
      info.numSmaller = ns[0];
      free(ns);
      return i;
    }
  }
  printf("OPPS\n");
  return 5367;
}

template<typename T,uint BIT_SHIFT>
void updateAnswerSoFar(T &answerSoFar,T digitValue){
  T digitMask = digitValue << BIT_SHIFT;
  answerSoFar |= digitMask;
}


template<typename T,uint RADIX_SIZE, uint BIT_SHIFT>
void  digitPass(uint *d_vec, uint &answerSoFar, uint *digitCounts,problemInfo_t &info){

  T currentDigit;
  uint neededSharedMemory = ((1 << RADIX_SIZE) ) * info.threadsPerBlock * sizeof(ushort);

  if(BIT_SHIFT == 28){
    getCounts<T,RADIX_SIZE,BIT_SHIFT><<<info.blocks, info.threadsPerBlock, neededSharedMemory>>>(d_vec, info.currentSize,digitCounts,info.totalThreads);
  }
  else{
    getCounts2<T,RADIX_SIZE,BIT_SHIFT><<<info.blocks, info.threadsPerBlock, neededSharedMemory>>>(d_vec, info.currentSize,digitCounts,info.totalThreads,answerSoFar);
  }

  currentDigit = determineDigit<T,RADIX_SIZE, BIT_SHIFT,0>(digitCounts, info);
 
  updateAnswerSoFar<T,BIT_SHIFT>(answerSoFar, currentDigit);
  info.previousDigit = currentDigit;
} 

template<typename T,uint RADIX_SIZE, uint BIT_SHIFT>
void  digitPassWithShrink(uint *d_vec, uint *alt_vec, uint &answerSoFar, uint *digitCounts, problemInfo_t &info){
  T currentDigit;
  uint neededSharedMemory = ((1 << RADIX_SIZE) ) * info.threadsPerBlock * sizeof(ushort);


  getCountsWithShrink<T,RADIX_SIZE,BIT_SHIFT><<<info.blocks, info.threadsPerBlock, neededSharedMemory>>>(d_vec,alt_vec,
                                                                                                         info.currentSize,
                                                                                                         digitCounts,
                                                                                                         info.totalThreads,
                                                                                                         info.numSmaller,
                                                                                                         info.previousDigit,
                                                                                                         answerSoFar);

  currentDigit = determineDigit<T,RADIX_SIZE, BIT_SHIFT,1>(digitCounts, info);
  

  updateAnswerSoFar<T,BIT_SHIFT>(answerSoFar, currentDigit);
  info.previousDigit = currentDigit;
} 

  void setupInfo(problemInfo_t &info, uint blocks, uint threadsPerBlock, uint size, uint k){
    info.blocks = blocks;
    info.threadsPerBlock = threadsPerBlock;
    info.totalThreads = blocks * threadsPerBlock;
    info.currentSize = size;
    info.nextSize = size; 
    info.k = k;
    info.countSize = (16 * blocks * threadsPerBlock) + 1;
    info.numSmaller = 0;
    info.previousDigit = 0;

  }

uint runDigitPasses(uint *d_vec, uint size, uint k, uint blocks, uint threadsPerBlock){
  uint answerSoFar = 0;
  uint *digitCounts, *altVector;
  problemInfo_t info;
  setupInfo(info,blocks, threadsPerBlock, size, k);
    hipMalloc(&digitCounts, info.countSize * sizeof(uint));

    digitPass<uint,4,28,PreprocessKeyFunctor<K> >(d_vec, answerSoFar, digitCounts,info);
      digitPass<uint,4,24,PreprocessKeyFunctor<K> >(d_vec, answerSoFar, digitCounts,info);
  hipMalloc(&altVector, info.nextSize * sizeof(uint));
  digitPassWithShrink<uint,4,20, PreprocessKeyFunctor<K> >(d_vec,altVector, answerSoFar, digitCounts, info);  
    digitPass<uint,4,16,PreprocessKeyFunctor<K> >(altVector, answerSoFar, digitCounts,info);
    digitPassWithShrink<uint,4,12,PreprocessKeyFunctor<K> >(altVector,d_vec,answerSoFar, digitCounts,info);
    digitPass<uint,4,8,PreprocessKeyFunctor<K> >(d_vec, answerSoFar, digitCounts,info);
    digitPassWithShrink<uint,4,4, PreprocessKeyFunctor<K> >(d_vec,altVector, answerSoFar, digitCounts,info);
    digitPass<uint,4,0,PreprocessKeyFunctor<K> >(altVector, answerSoFar, digitCounts,info);

  hipFree(altVector);
  hipFree(digitCounts);
  return answerSoFar;
}
template<typename T>
T radixSelect(T *d_vec, uint size, uint k){
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp,0) ;
  uint blocks = dp.multiProcessorCount ;//16;
  uint threadsPerBlock = dp.maxThreadsPerBlock;///2;
  return runDigitPasses(d_vec, size, k, blocks, threadsPerBlock);
}

}
