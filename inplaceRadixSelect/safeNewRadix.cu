#include "hip/hip_runtime.h"

#include <thrust/scan.h>
  #include <thrust/count.h>
  #include <thrust/reduce.h>
  #include <thrust/for_each.h>
#include "radixsort_key_conversion.h"
namespace DestructiveRadixSelect{

struct problemInfo_t{
  uint *d_vec;
  uint blocks;
  uint threadsPerBlock;
  uint totalThreads;
  uint curSize;
  uint size;
  uint k;
  uint countSize;
  uint previousDigit;
  };   

template<uint BIT_SHIFT>
__global__ void getCounts(uint *d_vec,const uint size, uint *digitCounts,const uint offset,uint answerSoFar) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  __shared__ uint counts[32];
   counts[threadIdx.x] = 0;
  int i;
  uint count[32];// =0;
  for(i =idx;i < size; i += offset){
    count[0] += __popc(__ballot( (d_vec[i] >> BIT_SHIFT) == (answerSoFar >> BIT_SHIFT)));
  }

  if(! (threadIdx.x % 32)){
    digitCounts[blockIdx.x * 32 + (threadIdx.x / 32)] = count[0];
  }
}

  template<uint BIT_SHIFT>
  uint determineDigit(uint *digitCounts,problemInfo_t &info){
  thrust::device_ptr<uint>ptr(digitCounts);
  // PrintFunctions::printCudaArray(digitCounts, info.countSize);
  uint sum = thrust::reduce(ptr, ptr + info.countSize);
  // printf("BIT: %d\n", BIT_SHIFT + 1);
  // printf("SUM: %d\n", sum);
  // printf("K: %d\n", info.k);
  // printf("CUR SIZE: %d\n", info.curSize);
  if(sum >= info.k){
    info.curSize = sum;
    return 1;
  }
  
  info.curSize -= sum;
  info.k -= sum;
  return 0;

}

template<uint BIT_SHIFT>
void updateAnswerSoFar(uint &answerSoFar,uint digitValue){
  uint digitMask = digitValue << BIT_SHIFT;
  answerSoFar |= digitMask;
}


 template<uint BIT_SHIFT >
 void  digitPass(uint &answerSoFar, uint *digitCounts,problemInfo_t &info){
   
   hipMemset(&digitCounts, 0, info.countSize * sizeof(uint));
   uint onesMask = 0x1 << BIT_SHIFT;
   // PrintFunctions::printBinary(answerSoFar);
   // PrintFunctions::printBinary(answerSoFar | onesMask);
   getCounts<BIT_SHIFT><<<info.blocks, info.threadsPerBlock>>>(info.d_vec, info.size,digitCounts,info.totalThreads, answerSoFar | onesMask);
   info.previousDigit = determineDigit<BIT_SHIFT>(digitCounts,info);
   updateAnswerSoFar<BIT_SHIFT>(answerSoFar,info.previousDigit);
 }


void setupInfo(problemInfo_t &info, uint blocks, uint threadsPerBlock, uint size, uint k, uint *d_vec){
    info.blocks = blocks *4 ;
    info.threadsPerBlock = 1024;
    info.totalThreads = info.blocks * info.threadsPerBlock;
    info.size = size;
    info.curSize = size;
    info.k = k;
    info.countSize = info.blocks * (info.threadsPerBlock / 32);
    info.previousDigit = 0;
    info.d_vec = d_vec;
  }



template<typename T>
T runDigitPasses(T *d_vec, uint size, uint k, uint blocks, uint threadsPerBlock){

  uint *digitCounts;
  problemInfo_t info;
  uint answerSoFar = 0;
  setupInfo(info,blocks, threadsPerBlock, size, k,d_vec);
  hipMalloc(&digitCounts, info.countSize * sizeof(uint));

  digitPass<31>(answerSoFar,digitCounts, info);
  digitPass<30>(answerSoFar,digitCounts, info);
  digitPass<29>(answerSoFar,digitCounts, info);
  digitPass<28>(answerSoFar,digitCounts, info);
  digitPass<27>(answerSoFar,digitCounts, info);
  digitPass<26>(answerSoFar,digitCounts, info);
  digitPass<25>(answerSoFar,digitCounts, info);
  digitPass<24>(answerSoFar,digitCounts, info);
  digitPass<23>(answerSoFar,digitCounts, info);
  digitPass<22>(answerSoFar,digitCounts, info);
  digitPass<21>(answerSoFar,digitCounts, info);
  digitPass<20>(answerSoFar,digitCounts, info);
  digitPass<19>(answerSoFar,digitCounts, info);
  digitPass<18>(answerSoFar,digitCounts, info);
  digitPass<17>(answerSoFar,digitCounts, info);
  digitPass<16>(answerSoFar,digitCounts, info);
  digitPass<15>(answerSoFar,digitCounts, info);
  digitPass<14>(answerSoFar,digitCounts, info);
  digitPass<13>(answerSoFar,digitCounts, info);
  digitPass<12>(answerSoFar,digitCounts, info);
  digitPass<11>(answerSoFar,digitCounts, info);
  digitPass<10>(answerSoFar,digitCounts, info);
  digitPass<9>(answerSoFar,digitCounts, info);
  digitPass<8>(answerSoFar,digitCounts, info);
  digitPass<7>(answerSoFar,digitCounts, info);
  digitPass<6>(answerSoFar,digitCounts, info);
  digitPass<5>(answerSoFar,digitCounts, info);
  digitPass<4>(answerSoFar,digitCounts, info);
  digitPass<3>(answerSoFar,digitCounts, info);
  digitPass<2>(answerSoFar,digitCounts, info);
  digitPass<1>(answerSoFar,digitCounts, info);
  digitPass<0>(answerSoFar,digitCounts, info);




  return answerSoFar;

}

uint radixSelect(uint *d_vec, uint size, uint k){
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp,0) ;
  uint blocks = dp.multiProcessorCount;
  uint threadsPerBlock = dp.maxThreadsPerBlock;
  return runDigitPasses(d_vec, size, k, blocks, threadsPerBlock);
}


}
