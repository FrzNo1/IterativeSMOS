/* Copyright 2011 Russel Steinbach, Jeffrey Blanchard, Bradley Gordon,
 *   and Toluwaloju Alabi
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
template <typename T>
 struct results_t{
  float time;
  T val;
};

template<typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T* h_vec, uint size, results_t<T> **result){
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *)malloc(sizeof(results_t<T>));
}

template<typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, T* d_vec, results_t<T> *result, float time, T value){
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->val = value;
  result->time = time;
  // hipDeviceSynchronize();
}
 
/////////////////////////////////////////////////////////////////
//          THE SORT AND CHOOSE TIMING FUNCTION
/////////////////////////////////////////////////////////////////
template<typename T>
results_t<T>* timeSortAndChoose(T *h_vec, uint numElements, uint k){

  
  T* d_vec;
  T returnValueFromSelect;
  results_t<T> *result;
  float time;
  hipEvent_t start, stop;
 
  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);

  thrust::device_ptr<T> dev_ptr(d_vec);
  hipEventRecord(start, 0);

  thrust::sort(dev_ptr, dev_ptr + numElements);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  hipMemcpy(h_vec, d_vec, numElements * sizeof(T), hipMemcpyDeviceToHost);
  returnValueFromSelect = h_vec[numElements - k];

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);
  return result;
}
template<typename T>
results_t<T>* timeInplaceRadixSelect(T *h_vec, uint numElements, uint k){


  float time;
  hipEvent_t start,stop;
  results_t<T> *result;
  T returnValueFromSelect;
  T *d_vec;

  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);
  thrust::device_ptr<T> dev_ptr(d_vec);

  hipEventRecord(start,0);
  //CALL THE WRAPPER FUNCTION
  returnValueFromSelect =  InplaceRadix::inplaceRadixSelectWrapper(d_vec,numElements,k);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);

  return result;
}

template<typename T>
results_t<T>* timeNewInplaceRadixSelect(T *h_vec, uint numElements, uint k){


  float time;
  hipEvent_t start,stop;
  results_t<T> *result;
  T returnValueFromSelect;
  T *d_vec;

  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);
  thrust::device_ptr<T> dev_ptr(d_vec);

  hipEventRecord(start,0);
  //CALL THE WRAPPER FUNCTION
  returnValueFromSelect =  NewInplaceRadixSelect::countingRadixSelect(d_vec,numElements,k);
    
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);

  return result;
}

template<typename T>
results_t<T>* timeDestructiveRadixSelect(T *h_vec, uint numElements, uint k){

  float time;
  hipEvent_t start,stop;
  results_t<T> *result;
  T returnValueFromSelect;
  T *d_vec;

  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);
  thrust::device_ptr<T> dev_ptr(d_vec);

  hipEventRecord(start,0);
  //CALL THE WRAPPER FUNCTION
  returnValueFromSelect =  DestructiveRadixSelect::radixSelect(d_vec,numElements,k);
    
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);

  wrapupForTiming(start,stop, d_vec, result, time,returnValueFromSelect);

  return result;
}
