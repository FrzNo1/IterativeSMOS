#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <limits>
#include <math.h>
#include <sys/time.h>
#include <time.h>

namespace IterativeSMOS {
    using namespace std;


#define SAFE
#define MAX_THREADS_PER_BLOCK 1024
#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

    /// ***********************************************************
    /// ***********************************************************
    /// **** HELPER SAFE FUNCTIONS
    /// ***********************************************************
    /// ***********************************************************
    // ************** SAFETY ERROR CHECK FUNCTIONS *************


    void check_malloc_int(int *pointer, const char *message)
    {
        if ( pointer == NULL ) {
            printf("Malloc failed for %s.\n", message);
        }
    }

    void check_malloc_float(float *pointer, const char *message)
    {
        if ( pointer == NULL ) {
            printf("Malloc failed for %s.\n", message);
        }
    }

    void check_malloc_double(double *pointer, const char *message)
    {
        if ( pointer == NULL ) {
            printf("Malloc failed for %s.\n", message);
        }
    }

    void check_cudaMalloc(const char *message)
    {
        hipError_t status = hipGetLastError();
        if (status != hipSuccess) {
            fprintf(stderr, "Error: hipMalloc failed for %s: %d\n", message, status);
        }
    }

    void Check_CUDA_Error(const char *message)
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "Error: %s: %s\n", message, hipGetErrorString(error) );
            exit(-1);
        }
    }

    /*
    void Check_CUBLAS_Error(const char *message)
    {
        hipblasStatus_t status = cublasGetError();
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf (stderr, "Error: %s: %d\n", message, status);
            exit(-1);
        }
    }
     */

    /*

    void Check_CURAND_Error(hiprandStatus_t curandCheck, const char *message)
    {
        if (curandCheck != HIPRAND_STATUS_SUCCESS) {
            fprintf (stderr, "Error: %s: %d\n", message, curandCheck);
            exit(-1);
        }
    }

     */



    void check_cudaMalloc2(hipError_t status, const char *message)
    {
        if (status != hipSuccess) {
            fprintf(stderr, "Error: hipMalloc failed for %s: %d\n", message, status);
        }
    }




// ************** SAFETY ERROR CHECK WRAPPERS TO BE USED IN CODE *************



    inline void SAFEcudaMalloc2(hipError_t status, const char *message)
    {
#ifdef SAFE
        check_cudaMalloc2(status, message);
#endif
    }



    inline void SAFEcudaMalloc(const char *message)
    {
#ifdef SAFE
        check_cudaMalloc(message);
#endif
    }

    inline void SAFEcuda(const char *message)
    {
#ifdef SAFE
        Check_CUDA_Error(message);
#endif
    }

    /*
    inline void SAFEcublas(const char *message)
    {
#ifdef SAFE
        Check_CUBLAS_Error(message);
#endif
    }
     */

    /*

    inline void SAFEcurand(hiprandStatus_t curandCheck, const char *message)
    {
#ifdef SAFE
        Check_CURAND_Error(curandCheck, message);
#endif
    }

     */

    inline void SAFEmalloc_int(int * pointer, const char *message)
    {
#ifdef SAFE
        check_malloc_int(pointer, message);
#endif
    }

    inline void SAFEmalloc_float(float *pointer, const char *message)
    {
#ifdef SAFE
        check_malloc_float(pointer, message);
#endif
    }

    inline void SAFEmalloc_double(double *pointer, const char *message)
    {
#ifdef SAFE
        check_malloc_double(pointer, message);
#endif
    }




    /// ***********************************************************
    /// ***********************************************************
    /// **** HELPER CPU FUNCTIONS
    /// ***********************************************************
    /// ***********************************************************

    /* This function initializes a vector to all zeros on the host (CPU).
     */
    template<typename T>
    void setToAllZero (T * d_vector, int length) {
        hipMemset(d_vector, 0, length * sizeof(T));
    }


    /* This function finds the bin containing the kth element we are looking for (works on
       the host). While doing the scan, it stores the sum-so-far of the number of elements in
       the buckets where k values fall into.

       markedBuckets : buckets containing the corresponding k values
       sums : sum-so-far of the number of elements in the buckets where k values fall into
    */
    inline int findKBuckets(unsigned int * d_bucketCount, unsigned int * h_bucketCount, int numBuckets
            , const unsigned int * kVals, int numKs, unsigned int * sums, unsigned int * markedBuckets
            , int numBlocks) {
        // consider the last row which holds the total counts
        int sumsRowIndex= numBuckets * (numBlocks-1);

        hipMemcpy(h_bucketCount, d_bucketCount + sumsRowIndex,
                   sizeof(unsigned int) * numBuckets, hipMemcpyDeviceToHost);

        int kBucket = 0;
        int k;
        int sum = h_bucketCount[0];

        for(int i = 0; i < numKs; i++) {
            k = kVals[i];
            while ((sum < k) & (kBucket < numBuckets - 1)) {
                kBucket++;
                sum += h_bucketCount[kBucket];
            }
            markedBuckets[i] = kBucket;
            sums[i] = sum - h_bucketCount[kBucket];
        }

        return 0;
    }

    /*
     * This function updates the correct kth orderstats if the bin only contains one element. While going through the
     * list of orderstats, it updates K since we have reduced the problem size to elements in the kth bucket. In
     * addition, it updates the unique buckets list to avoid the situation where two order share the same buckets.
     *
     * kthBucketScanner:  sum-so-far of the number of elements in the buckets where k values fall into
     * uniqueBuckets:  the list to store all buckets which are active with no repeats
     * tempKorderBucket:  buckets which have only one element. That is, the bucket with correct kth orderstats
     */
    template <typename T>
    inline int updatekVals_iterative(unsigned int * kVals, int * numKs, T * output, unsigned int * kIndicies,
                                     int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
                                     unsigned int * kthBucketScanner, unsigned int * reindexCounter,
                                     unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
                                     int * numUniqueBuckets, int * numUniqueBucketsOld,
                                     unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength) {
        int index = 0;
        int numKsindex = 0;
        *numUniqueBucketsOld = *numUniqueBuckets;
        *numUniqueBuckets = 0;
        *lengthOld = *length;
        *tempKorderLength = 0;

        // go through the markedbucket list. If there is only one element in array, we update it to tempKorderBucket
        while (index < *numKs) {
            if (h_bucketCount[markedBuckets[index]] == 1) {
                tempKorderIndeces[*tempKorderLength] = kIndicies[index];
                tempKorderBucket[*tempKorderLength] = markedBuckets[index];
                (*tempKorderLength)++;
                index++;
                continue;
            }

            break;
        }

        // get the index of the first buckets with more than one elements in it
        // add the number of elements and updates correct kth order
        if (index < *numKs) {
            uniqueBuckets[0] = markedBuckets[index];
            uniqueBucketCounts[0] = h_bucketCount[markedBuckets[index]];
            reindexCounter[0] = 0;
            *numUniqueBuckets = 1;
            kVals[0] = kVals[index] - kthBucketScanner[index];
            kIndicies[0] = kIndicies[index];
            numKsindex++;
            index++;
        }

        // go through the markedbuckets list. If there is only one element in that bucket, updates it to
        // tempKorderBucket; if there is more than one, updates it to uniqueBucket
        for ( ; index < *numKs; index++) {

            // case if there is only one element
            if (h_bucketCount[markedBuckets[index]] == 1) {
                tempKorderIndeces[*tempKorderLength] = kIndicies[index];
                tempKorderBucket[*tempKorderLength] = markedBuckets[index];
                (*tempKorderLength)++;
                continue;
            }

            // case if the there is more than one element in the bucket and the bucket is not repeat with last one
            if (markedBuckets[index] != uniqueBuckets[(*numUniqueBuckets) - 1]) {
                uniqueBuckets[*numUniqueBuckets] = markedBuckets[index];
                uniqueBucketCounts[*numUniqueBuckets] = h_bucketCount[markedBuckets[index]];
                reindexCounter[*numUniqueBuckets] = reindexCounter[(*numUniqueBuckets) - 1]
                                                    + uniqueBucketCounts[(*numUniqueBuckets) - 1];
                (*numUniqueBuckets)++;
            }

            // update korder
            kVals[numKsindex] = reindexCounter[(*numUniqueBuckets) - 1] + kVals[index] - kthBucketScanner[index];
            kIndicies[numKsindex] = kIndicies[index];
            numKsindex++;
        }

        // update numKs and length of vector
        *numKs = numKsindex;
        if (*numKs > 0)
            *length = reindexCounter[(*numUniqueBuckets) - 1] + uniqueBucketCounts[(*numUniqueBuckets) - 1];


        return 0;
    }

    /*
     * This function swap pointers for the two lists
     */
    template <typename T>
    void swapPointers(T** a, T** b) {
        T * temp = * a;
        * a = * b;
        * b = temp;
    }



    /// ***********************************************************
    /// ***********************************************************
    /// **** HELPER GPU FUNCTIONS-KERNELS
    /// ***********************************************************
    /// ***********************************************************


    /*
     * This function generate new buckets offset and slopes by giving the new pivots and number of elements in
     * that buckets
     *
     * pivotsLeft & pivotsRight:  the bounds of elements for each bucket
     * kthnumBuckets:  array to store bucket offset.
     */
    template <typename T>
    __global__ void generateBucketsandSlopes_iterative (T * pivotsLeft, T * pivotsRight, double * slopes,
                                                        unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
                                                        unsigned int * kthnumBuckets, int length, int offset, int numBuckets) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

        // Assign bucket number and slope to first to the second to last active buckets
        if (index < numUniqueBuckets - 1) {
            for (int i = index; i < numUniqueBuckets - 1; i += offset) {

                // assign bucket number
                kthnumBuckets[i] = max(uniqueBucketsCounts[i] * numBuckets / length ,2);

                // assign slope
                slopes[i] = (double) kthnumBuckets[i] / (double) (pivotsRight[i] - pivotsLeft[i]);
            }
        }

        __syncthreads();

        // Assign bucket number and slope to the last active buckets
        if (index < 1) {
            // exclusive cumulative sum to the kthnumbuckets for finding the correct number of buckets
            // for the last active buckets
            // thrust::exclusive_scan(thrust::device, kthnumBuckets, kthnumBuckets + numUniqueBuckets, kthnumBuckets, 0);

            if (numUniqueBuckets > 1) {
                for (int i = 1; i < numUniqueBuckets; i++) {
                    kthnumBuckets[i] += kthnumBuckets[i - 1];
                }

                for (int i = numUniqueBuckets - 1; i > 0; i--) {
                    kthnumBuckets[i] = kthnumBuckets[i - 1];
                }
            }

            kthnumBuckets[0] = 0;
            // bucket number is assigned automatically

            // assign slope
            slopes[numUniqueBuckets - 1] = (numBuckets - kthnumBuckets[numUniqueBuckets - 1])
                                           / (double) (pivotsRight[numUniqueBuckets - 1] - pivotsLeft[numUniqueBuckets - 1]);
        }
    }


    /* This function assigns elements to buckets based on the pivots and slopes determined
       by a randomized sampling of the elements in the vector. At the same time, this
       function keeps track of count.

       d_elementToBucket : bucket assignment for every array element
       d_bucketCount : number of element that falls into the indexed buckets within the block
    */
    template <typename T>
    __global__ void assignSmartBucket_iterative(T * d_vector, int length, unsigned int * d_elementToBucket,
                                                double * slopes, T * pivotsLeft, T * pivotsRight,
                                                unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
                                                int numUniqueBuckets, int numBuckets, int offset) {

        int index = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int bucketIndex;
        int threadIndex = threadIdx.x;

        /*
        if (index == 1) {
            printf("assignmentSmartBucket in kernel\n\n");
        }
         */


      

        //variables in shared memory for fast access
        extern __shared__ unsigned int array[];
        double * sharedSlopes = (double *)array;
        T * sharedPivotsLeft = (T *)&sharedSlopes[numUniqueBuckets];
        unsigned int * sharedkthNumBuckets = (unsigned int *)&sharedPivotsLeft[numUniqueBuckets];
        unsigned int * sharedBuckets = (unsigned int *)&sharedkthNumBuckets[numUniqueBuckets];

        if (index < 1) {
             printf("executed to phase 0\n\n");
             /*
             for (int bb=0; bb < 16; bb++) {
               printf("bb=%d, vec=%d, elemtobuck=%d, slopes=%lf buckCout=%d, pleft=%d, pright=%d \n ", bb, d_vector[bb], d_elementToBucket[bb],slopes[bb], d_bucketCount[bb], pivotsLeft[bb], pivotsRight[bb]);
             }
             printf("\n \n \n");
             */
        }

        //reading bucket counts into shared memory where increments will be performed
        for (int i = 0; i < (numBuckets / MAX_THREADS_PER_BLOCK); i++) {

            if (threadIndex < numBuckets)
                sharedBuckets[i * MAX_THREADS_PER_BLOCK + threadIndex] = 0;
        }
  //    if (index < length) {
        if (index < 1)
            printf("executed to phase 1\n\n");


        if (threadIndex < numUniqueBuckets) {
            sharedPivotsLeft[threadIndex] = pivotsLeft[threadIndex];
            sharedSlopes[threadIndex] = slopes[threadIndex];
            sharedkthNumBuckets[threadIndex] = kthNumBuckets[threadIndex];
            //printf("PL=%d, Slps=%lf, kNumB=%d \n", pivotsLeft[threadIndex], slopes[threadIndex], kthNumBuckets[threadIndex]);
            //printf("sPL=%d, sSlps=%lf, skNumB=%d \n", sharedPivotsLeft[threadIndex], sharedSlopes[threadIndex], sharedkthNumBuckets[threadIndex]);
        }

        if (index < 1)
            printf("executed to phase 2\n\n");

 //       if (index < length)
 //         printf("index=%d, length=%d, numUniqueBuckets=%d, offset=%d \n", index, length, numUniqueBuckets, offset);
          
        __syncthreads();

        if (index < 1)
            printf("executed to phase 3\n\n");


        //assigning elements to buckets and incrementing the bucket counts
        if (index < length) {
            for (int i = index; i < length; i += offset) {
                T num = d_vector[i];
                int pivotIndex;

                // potential to change it to binary search to be faster
                if (num >= sharedPivotsLeft[numUniqueBuckets - 1]) {
                    pivotIndex = numUniqueBuckets - 1;
                }
                else {
                    for (int j = 0; j < numUniqueBuckets - 1; j++) {
                        if (num >= sharedPivotsLeft[j] && num <= sharedPivotsLeft[j + 1]) {
                            pivotIndex = j;
                            break;
                        }
                    }
                }

                /*
                int minPivotIndex = 0;
                int maxPivotIndex = numUniqueBuckets - 1;
                int midPivotIndex;

                // find the index of left pivots that is greatest s.t. lower than or equal to
                // num using binary search
                if (num >= pivotsLeft[numUniqueBuckets - 1]) {
                    minPivotIndex = numUniqueBuckets - 1;
                }
                else {
                    for (int j = 1; j < numUniqueBuckets - 1; j *= 2) {
                        midPivotIndex = (maxPivotIndex + minPivotIndex) / 2;
                        if (num >= pivotsLeft[midPivotIndex])
                            minPivotIndex = midPivotIndex;
                        else
                            maxPivotIndex = midPivotIndex;
                    }
                }
                 */

                bucketIndex = sharedkthNumBuckets[pivotIndex]
                              + (int) (((double)num - (double)sharedPivotsLeft[pivotIndex])
                                       * sharedSlopes[pivotIndex]);


                // potential to remove the for loop
                if (sharedPivotsLeft[pivotIndex] != pivotsRight[pivotIndex]) {
                    if (bucketIndex == numBuckets) {
                        bucketIndex--;
                    }
                    else if (pivotIndex < numUniqueBuckets - 1) {
                        if (bucketIndex == sharedkthNumBuckets[pivotIndex + 1]) {
                            bucketIndex--;
                        }
                    }
                }

                d_elementToBucket[i] = bucketIndex;
                atomicInc(sharedBuckets + bucketIndex, length);

   //             printf("%d, %d;  ", d_vector[i], d_elementToBucket[i]);
            }
        }

        if (index < 1)
            printf("executed to phase 4\n\n");
            
            
   //    } // closes the if (index < max(length))

        __syncthreads();

        //reading bucket counts from shared memory back to global memory
        for (int i = 0; i <(numBuckets / MAX_THREADS_PER_BLOCK); i++)
            if (threadIndex < numBuckets)
                *(d_bucketCount + blockIdx.x * numBuckets
                  + i * MAX_THREADS_PER_BLOCK + threadIndex) =
                        *(sharedBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex);

        if (index < 1)
            printf("executed to phase 5\n\n");
            
    } // closes the kernel

    /* This function cumulatively sums the count of every block for a given bucket s.t. the
       last block index holds the total number of elements falling into that bucket all over the
       array.
       updates d_bucketCount
    */
    __global__ void sumCounts(unsigned int * d_bucketCount, const int numBuckets
            , const int numBlocks) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        for(int j=1; j<numBlocks; j++)
            d_bucketCount[index + numBuckets*j] += d_bucketCount[index + numBuckets*(j-1)];
    }


    /* This function reindexes the buckets counts for every block according to the
       accumulated d_reindexCounter counter for the reduced vector.
       updates d_bucketCount
    */
    __global__ void reindexCounts(unsigned int * d_bucketCount, int numBuckets, int numBlocks,
                                  unsigned int * d_reindexCounter, unsigned int * d_uniqueBuckets,
                                  const int numUniqueBuckets) {
        int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadIndex < numUniqueBuckets) {
            int index = d_uniqueBuckets[threadIndex];
            unsigned int add = d_reindexCounter[threadIndex];

            for (int j = 0; j < numBlocks; j++)
                d_bucketCount[index + numBuckets * j] += add;
        }
    }

    /* This function copies the elements of buckets that contain kVals into a newly allocated
       reduced vector space.
       newArray - reduced size vector containing the essential elements
    */
    template <typename T>
    __global__ void copyElements_iterative (T * d_vector, T * d_newvector, int lengthOld, unsigned int * elementToBuckets,
                                            unsigned int * uniqueBuckets, int numUniqueBuckets,
                                            unsigned int * d_bucketCount, int numBuckets, unsigned int offset) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int threadIndex;
        int loop = numBuckets / MAX_THREADS_PER_BLOCK;

        extern __shared__ unsigned int sharedBuckets[];

        for (int i = 0; i <= loop; i++) {
            threadIndex = i * blockDim.x + threadIdx.x;
            if (threadIndex < numUniqueBuckets)
                sharedBuckets[threadIndex] = uniqueBuckets[threadIndex];
        }

        __syncthreads();


        if (index < lengthOld) {

            for (int i = index; i < lengthOld; i += offset) {
                unsigned int temp = elementToBuckets[i];

                for (int j = 0; j < numUniqueBuckets; j++) {
                    if (temp == sharedBuckets[j]) {
                        d_newvector[atomicDec(d_bucketCount + blockIdx.x * numBuckets
                                              + sharedBuckets[j], lengthOld) - 1] = d_vector[i];
                        break;
                    }
                }


                /*
                int minBucketIndex = 0;
                int maxBucketIndex = numUniqueBuckets - 1;
                int midBucketIndex;

                for (int j = 1; j < numUniqueBuckets; j *= 2) {
                    midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
                    if (temp > sharedBuckets[midBucketIndex])
                        minBucketIndex = midBucketIndex + 1;
                    else
                        maxBucketIndex = midBucketIndex;
                }

                if (temp == sharedBuckets[maxBucketIndex])
                    d_newvector[atomicDec(d_bucketCount + blockIdx.x * numBuckets
                                          + sharedBuckets[maxBucketIndex], lengthOld) - 1] = d_vector[i];
                */
            }
        }

        // needs to swap d_vector with d_newvector
    }


    /* This function copies the elements of buckets that contain kVals into a newly allocated
       reduced vector space.
       newArray - reduced size vector containing the essential elements
    */
    template <typename T>
    __global__ void updatePivots_iterative(T * d_pivotsLeft, T * d_newPivotsLeft, T * d_newPivotsRight,
                                           double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
                                           int numUniqueBuckets, int numUniqueBucketsOld, int offset) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < numUniqueBuckets) {
            for (int i = index; i < numUniqueBuckets; i += offset) {

                unsigned int bucket = uniqueBuckets[i];
                unsigned int bucketIndex;

                // potential to change to binary search to make it faster
                if (bucket >= kthnumBuckets[numUniqueBucketsOld - 1]) {
                    bucketIndex = numUniqueBucketsOld - 1;
                }
                else {
                    for (int j = 0; j < numUniqueBucketsOld - 1; j++) {
                        if (bucket >= kthnumBuckets[j] && bucket < kthnumBuckets[j + 1]) {
                            bucketIndex = j;
                            break;
                        }
                    }
                }


                /*
                // // perform binary search to find kthNumBucket that is greatest s.t. lower than or equal to the bucket
                int minBucketIndex = 0;
                int maxBucketIndex = numUniqueBucketsOld - 1;
                int midBucketIndex;

                if (bucket >= kthnumBuckets[numUniqueBucketsOld - 1]) {
                    minBucketIndex = numUniqueBucketsOld - 1;
                }
                else {
                    for (int j = 1; j < numUniqueBucketsOld - 1; j *= 2) {
                        midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
                        if (bucket >= kthnumBuckets[midBucketIndex])
                            minBucketIndex = midBucketIndex;
                        else
                            maxBucketIndex = midBucketIndex;
                    }
                }
                */

                if (slopes[bucketIndex] == (double)0) {
                    d_newPivotsLeft[i] = d_pivotsLeft[bucketIndex];
                    d_newPivotsRight[i] = d_pivotsLeft[bucketIndex];
                }
                else {
                    d_newPivotsLeft[i] = d_pivotsLeft[bucketIndex] +
                                         (T) (((double) (bucket - kthnumBuckets[bucketIndex])) / slopes[bucketIndex]);
                    d_newPivotsRight[i] = d_pivotsLeft[bucketIndex] +
                                          (T) (((double) (bucket - kthnumBuckets[bucketIndex] + 1)) /
                                               slopes[bucketIndex]);
                }
            }
        }

        // needs to swap pointers of pivotsLeft with newPivotsLeft, pivotsRight with newPivotsRight
    }

    /*
     * This function finds the actual element for the kth orderstats by giving the list of buckets
     */
    template <typename T>
    __global__ void updateOutput_iterative (T * d_vector, unsigned int * d_elementToBucket, int lengthOld, T * d_tempOutput,
                                            unsigned int * d_tempKorderBucket, int tempKorderLength, int offset){

        int index = blockDim.x * blockIdx.x + threadIdx.x;

        if (index < lengthOld) {
            for (int i = index; i < lengthOld; i += offset) {
                unsigned int bucket = d_elementToBucket[i];

                for (int j = 0; j < tempKorderLength; j++) {
                    if (d_tempKorderBucket[j] == bucket)
                        d_tempOutput[j] = d_vector[i];
                }
            }
        }
    }




    /// ***********************************************************
    /// ***********************************************************
    /// **** GENERATE KD PIVOTS
    /// ***********************************************************
    /// ***********************************************************

    /* Hash function using Monte Carlo method
     */
    __host__ __device__
    unsigned int hash(unsigned int a) {
        a = (a+0x7ed55d16) + (a<<12);
        a = (a^0xc761c23c) ^ (a>>19);
        a = (a+0x165667b1) + (a<<5);
        a = (a+0xd3a2646c) ^ (a<<9);
        a = (a+0xfd7046c5) + (a<<3);
        a = (a^0xb55a4f09) ^ (a>>16);
        return a;
    }



    /* RandomNumberFunctor
     */
    struct RandomNumberFunctor :
            public thrust::unary_function<unsigned int, float> {
        unsigned int mainSeed;

        RandomNumberFunctor(unsigned int _mainSeed) :
                mainSeed(_mainSeed) {}

        __host__ __device__
        float operator()(unsigned int threadIdx)
        {
            unsigned int seed = hash(threadIdx) * mainSeed;

            thrust::default_random_engine rng(seed);
            rng.discard(threadIdx);
            thrust::uniform_real_distribution<float> u(0, 1);

            return u(rng);
        }
    };



    /* This function creates a random vector of 1024 elements in the range [0 1]
     */
    template <typename T>
    void createRandomVector(T * d_vec, int size) {
        timeval t1;
        unsigned int seed;

        gettimeofday(&t1, NULL);
        seed = t1.tv_usec * t1.tv_sec;
        // seed = 1000000000;

        thrust::device_ptr<T> d_ptr(d_vec);
        thrust::transform (thrust::counting_iterator<unsigned int>(0),
                           thrust::counting_iterator<unsigned int>(size),
                           d_ptr, RandomNumberFunctor(seed));
    }



    /* This function maps the [0 1] range to the [0 vectorSize] and
       grabs the corresponding elements.
    */
    template <typename T>
    __global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
        *(in + blockIdx.x*blockDim.x + threadIdx.x) =
                *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
    }

    __global__ void enlargeIndexAndGetElements (float * in, int * out, int * list, int size) {
        *(out + blockIdx.x * blockDim.x + threadIdx.x) =
                (int) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
    }


    __global__ void enlargeIndexAndGetElements (float * in, unsigned int * out, unsigned int * list, int size) {
        *(out + blockIdx.x * blockDim.x + threadIdx.x) =
                (unsigned int) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
    }



    /* This function generates Pivots from the random sampled data and calculates slopes.

       pivots - arrays of pivots
       slopes - array of slopes
    */
    template <typename T>
    void generatePivots (int * pivots, double * slopes, int * d_list, int sizeOfVector
            , int numPivots, int sizeOfSample, int totalSmallBuckets, int min, int max) {

        float * d_randomFloats;
        int * d_randomInts;
        int endOffset = 22;
        int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
        int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

        hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);

        d_randomInts = (int *) d_randomFloats;

        createRandomVector (d_randomFloats, sizeOfSample);

        // converts randoms floats into elements from necessary indices
        enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
        , MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list,
                                   sizeOfVector);



        pivots[0] = min;
        pivots[numPivots-1] = max;

        thrust::device_ptr<T>randoms_ptr(d_randomInts);
        thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

        hipDeviceSynchronize();

        // set the pivots which are next to the min and max pivots using the random element
        // endOffset away from the ends
        hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (int)
                , hipMemcpyDeviceToHost);
        hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1,
                    sizeof (int), hipMemcpyDeviceToHost);
        slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

        for (int i = 2; i < numPivots - 2; i++) {
            hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1,
                        sizeof (int), hipMemcpyDeviceToHost);
            slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
        }
        
        printf("\n\n\n\n%d %d %d %d %d\n\n\n\n", pivots[0], pivots[4], pivots[7], pivots[10], pivots[16]);

        slopes[numPivots - 3] = numSmallBuckets /
                                (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
        slopes[numPivots - 2] = numSmallBuckets /
                                (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

        hipFree(d_randomFloats);
    }





    /* This function generates Pivots from the random sampled data and calculates slopes.

       pivots - arrays of pivots
       slopes - array of slopes
    */
    template <typename T>
    void generatePivots (unsigned int * pivots, double * slopes, unsigned int * d_list, int sizeOfVector
            , int numPivots, int sizeOfSample, int totalSmallBuckets, unsigned int min, unsigned int max) {

        float * d_randomFloats;
        unsigned int * d_randomInts;
        int endOffset = 22;
        int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
        int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

        hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);

        d_randomInts = (unsigned int *) d_randomFloats;

        createRandomVector (d_randomFloats, sizeOfSample);

        // converts randoms floats into elements from necessary indices
        enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
        , MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list,
                                   sizeOfVector);



        pivots[0] = min;
        pivots[numPivots-1] = max;

        thrust::device_ptr<T>randoms_ptr(d_randomInts);
        thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

        hipDeviceSynchronize();

        // set the pivots which are next to the min and max pivots using the random element
        // endOffset away from the ends
        hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (unsigned int)
                , hipMemcpyDeviceToHost);
        hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1,
                    sizeof (unsigned int), hipMemcpyDeviceToHost);
        slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

        for (int i = 2; i < numPivots - 2; i++) {
            hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1,
                        sizeof (unsigned int), hipMemcpyDeviceToHost);
            slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
        }
        
        printf("\n\n\n\n%d %d %d %d %d\n\n\n\n", pivots[0], pivots[4], pivots[7], pivots[10], pivots[16]);

        slopes[numPivots - 3] = numSmallBuckets /
                                (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
        slopes[numPivots - 2] = numSmallBuckets /
                                (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

        hipFree(d_randomFloats);
    }

    template <typename T>
    void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector
            , int numPivots, int sizeOfSample, int totalSmallBuckets, T min, T max) {
        T * d_randoms;
        int endOffset = 22;
        int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
        int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

        hipMalloc (&d_randoms, sizeof (T) * sizeOfSample);

        createRandomVector (d_randoms, sizeOfSample);

        // converts randoms floats into elements from necessary indices
        enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
        , MAX_THREADS_PER_BLOCK>>>(d_randoms, d_list, sizeOfVector);

        pivots[0] = min;
        pivots[numPivots - 1] = max;

        thrust::device_ptr<T>randoms_ptr(d_randoms);
        thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

        hipDeviceSynchronize();

        // set the pivots which are endOffset away from the min and max pivots
        hipMemcpy (pivots + 1, d_randoms + endOffset - 1, sizeof (T),
                    hipMemcpyDeviceToHost);
        hipMemcpy (pivots + numPivots - 2, d_randoms + sizeOfSample - endOffset - 1,
                    sizeof (T), hipMemcpyDeviceToHost);
        slopes[0] = numSmallBuckets / ((double)pivots[1] - (double)pivots[0]);

        for (int i = 2; i < numPivots - 2; i++) {
            hipMemcpy (pivots + i, d_randoms + pivotOffset * (i - 1) + endOffset - 1,
                        sizeof (T), hipMemcpyDeviceToHost);
            slopes[i - 1] = numSmallBuckets / ((double) pivots[i] - (double) pivots[i - 1]);
        }

        slopes[numPivots - 3] = numSmallBuckets /
                                ((double)pivots[numPivots - 2] - (double)pivots[numPivots - 3]);
        slopes[numPivots - 2] = numSmallBuckets /
                                ((double)pivots[numPivots - 1] - (double)pivots[numPivots - 2]);

        hipFree(d_randoms);
    }




    /// ***********************************************************
    /// ***********************************************************
    /// **** iterativeSMOS: the main algorithm
    /// ***********************************************************
    /// ***********************************************************


    /* This function is the main process of the algorithm. It reduces the given multi-selection
       problem to a smaller problem by using bucketing ideas.
    */
    template <typename T>
    T iterativeSMOS (T* d_vector, int length, unsigned int * kVals, int numKs, T * output, int blocks
            , int threads, int numBuckets, int numPivots) {

        /// ***********************************************************
        /// **** STEP 1: Initialization
        /// **** STEP 1.1: Find Min and Max of the whole vector
        /// **** We don't need to go through the rest of the algorithm if it's flat
        /// ***********************************************************

        //find max and min with thrust
        T maximum, minimum;

        thrust::device_ptr<T>dev_ptr(d_vector);
        thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result =
                thrust::minmax_element(dev_ptr, dev_ptr + length);

        minimum = *result.first;
        maximum = *result.second;

        //if the max and the min are the same, then we are done
        if (maximum == minimum) {
            for (int i = 0; i < numKs; i++)
                output[i] = minimum;

            return 1;
        }

        SAFEcuda("End of STEP 1.1\n");


        /// ***********************************************************
        /// **** STEP 1: Initialization
        /// **** STEP 1.2: Declare variables and allocate memory
        /// **** Declare Variables
        /// ***********************************************************

        // declare variables for kernel launches
        int threadsPerBlock = threads;
        int numBlocks = blocks;
        int offset = blocks * threads;

        // variables for the randomized selection
        int sampleSize = 1024;

        // pivots variables
        // potential to simplify
        int numMemory;
        if (numKs > numPivots)
            numMemory = numKs;
        else
            numMemory = numPivots;  // replace this with max

        double * slopes = (double*)malloc(numMemory * sizeof(double));                  // size will be different
        double * d_slopes;
        T * pivots = (T*)malloc(numPivots * sizeof(T));
        T * d_pivots;
        CUDA_CALL(hipMalloc(&d_slopes, numMemory * sizeof(double)));
        CUDA_CALL(hipMalloc(&d_pivots, numPivots * sizeof(T)));

        T * pivotsLeft = (T*)malloc(numMemory * sizeof(T));                                 // new variables
        T * pivotsRight = (T*)malloc(numMemory * sizeof(T));
        T * d_pivotsLeft;
        T * d_pivotsRight;
        T * newPivotsLeft = (T*)malloc(numMemory * sizeof(T));                            // potential not being used
        T * newPivotsRight = (T*)malloc(numMemory * sizeof(T));                           // potential not being used
        T * d_newPivotsLeft;
        T * d_newPivotsRight;
        CUDA_CALL(hipMalloc(&d_pivotsLeft, numMemory * sizeof(T)));
        CUDA_CALL(hipMalloc(&d_pivotsRight, numMemory * sizeof(T)));
        CUDA_CALL(hipMalloc(&d_newPivotsLeft, numMemory * sizeof(T)));
        CUDA_CALL(hipMalloc(&d_newPivotsRight, numMemory * sizeof(T)));


        //Allocate memory to store bucket assignments
        size_t size = length * sizeof(unsigned int);
        unsigned int * d_elementToBucket;    //array showing what bucket every element is in
        CUDA_CALL(hipMalloc(&d_elementToBucket, size));


        // Allocate memory to store bucket counts
        size_t totalBucketSize = numBlocks * numBuckets * sizeof(unsigned int);
        unsigned int * h_bucketCount = (unsigned int *) malloc (numBuckets * sizeof (unsigned int));
        //array showing the number of elements in each bucket
        unsigned int * d_bucketCount;
        CUDA_CALL(hipMalloc(&d_bucketCount, totalBucketSize));

        // Allocate memory to store the new vector for kVals
        T * d_newvector;
        CUDA_CALL(hipMalloc(&d_newvector, length * sizeof(T)));


        // array of kth buckets
        int numUniqueBuckets;
        int numUniqueBucketsOld;
        int lengthOld;
        int tempKorderLength;
        unsigned int * d_kVals;
        unsigned int * kthBuckets = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_kthBuckets;
        unsigned int * kthBucketScanner = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_kthBucketScanner;
        unsigned int * kIndices = (unsigned int *)malloc(numKs * sizeof(unsigned int));
        unsigned int * d_kIndices;
        unsigned int * uniqueBuckets = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_uniqueBuckets;
        unsigned int * uniqueBucketCounts = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_uniqueBucketCounts;
        unsigned int * reindexCounter = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_reindexCounter;
        unsigned int * kthnumBuckets = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_kthnumBuckets;
        T * tempOutput = (T *)malloc(numMemory * sizeof(T));
        T * d_tempOutput;
        unsigned int * tempKorderBucket = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_tempKorderBucket;
        unsigned int * tempKorderIndeces = (unsigned int *)malloc(numMemory * sizeof(unsigned int));
        unsigned int * d_tempKorderIndeces;
        CUDA_CALL(hipMalloc(&d_kVals, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_kIndices, numKs * sizeof (unsigned int)));
        CUDA_CALL(hipMalloc(&d_kthBuckets, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_kthBucketScanner, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_uniqueBuckets, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_uniqueBucketCounts, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_reindexCounter, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_kthnumBuckets, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_tempOutput, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_tempKorderBucket, numMemory * sizeof(unsigned int)));
        CUDA_CALL(hipMalloc(&d_tempKorderIndeces, numMemory * sizeof(unsigned int)));

        for (int i = 0; i < numMemory; i++) {
            kthBucketScanner[i] = 0;
            // kIndices[i] = i;
        }

        for (int i = 0; i < numKs; i++) {
            kIndices[i] = i;
        }

        SAFEcuda("End of STEP 1.2\n");


        /// ***********************************************************
        /// **** STEP 1: Initialization
        /// **** STEP 1.3: Sort the klist
        /// **** and we have to keep the old index
        /// ***********************************************************

        CUDA_CALL(hipMemcpy(d_kIndices, kIndices, numKs * sizeof (unsigned int),
                             hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_kVals, kVals, numKs * sizeof (unsigned int),
                             hipMemcpyHostToDevice));

        // sort the given indices
        thrust::device_ptr<unsigned int>kVals_ptr(d_kVals);
        thrust::device_ptr<unsigned int>kIndices_ptr(d_kIndices);
        thrust::sort_by_key(kVals_ptr, kVals_ptr + numKs, kIndices_ptr);

        CUDA_CALL(hipMemcpy(kIndices, d_kIndices, numKs * sizeof (unsigned int),
                             hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(kVals, d_kVals, numKs * sizeof (unsigned int),
                             hipMemcpyDeviceToHost));

        /*
        int kMaxIndex = numKs - 1;
        int kOffsetMax = 0;
        while (kVals[kMaxIndex] == length) {
            output[kIndices[numKs-1]] = maximum;
            numKs--;
            kMaxIndex--;
            kOffsetMax++;
        }

        int kOffsetMin = 0;
        while (kVals[0] == 1) {
            output[kIndices[0]] = minimum;
            kIndices++;
            kVals++;
            numKs--;
            kOffsetMin++;
        }
         */



        //display information
        printf("Before entering the loop\n");
        printf("vector length: %d, kVals length: %d\n", length, numKs);
        printf("\n");

        SAFEcuda("End of STEP 1.3\n");




        /// ***********************************************************
        /// **** STEP 2: CreateBuckets
        /// ****  Declare and Generate Pivots and Slopes
        /// ***********************************************************
        
        // printf("%d, %d, %d, %d\n", pivots[0],pivots[7],pivots[11],pivots[16]);

        // Find bucket sizes using a randomized selection
        generatePivots<T>(pivots, slopes, d_vector, length, numPivots, sampleSize,
                          numBuckets, minimum, maximum);

        // printf("%d, %d, %d, %d\n", pivots[0],pivots[7],pivots[11],pivots[16]);

        // make any slopes that were infinity due to division by zero (due to no
        //  difference between the two associated pivots) into zero, so all the
        //  values which use that slope are projected into a single bucket
        for (int i = 0; i < numPivots - 1; i++)
            if (isinf(slopes[i]))
                slopes[i] = 0;



        // documentation
        for (int i = 0; i < numPivots - 1; i++) {
            pivotsLeft[i] = pivots[i];
            pivotsRight[i] = pivots[i + 1];
            kthnumBuckets[i] = numBuckets / (numPivots - 1) * i;
        }
        numUniqueBuckets = numPivots - 1;




        //display information
        printf("PivotsLeft: \n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", pivotsLeft[i]);
        printf("\n");
        printf("PivotsRight: \n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", pivotsRight[i]);
        printf("\n");
        printf("slopes: \n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%lf, ", slopes[i]);
        printf("\n");
        printf("kthnumBuckets: \n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", kthnumBuckets[i]);
        printf("\n");
        printf("\n");






        CUDA_CALL(hipMemcpy(d_slopes, slopes, (numPivots - 1) * sizeof(double),
                             hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_pivotsLeft, pivotsLeft, numUniqueBuckets * sizeof(T),
                             hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_pivotsRight, pivotsRight, numUniqueBuckets * sizeof(T),
                             hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_kthnumBuckets, kthnumBuckets, numUniqueBuckets * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

        SAFEcuda("End of STEP 2\n");



        /// ***********************************************************
        /// **** STEP 3: AssignBuckets
        /// **** Using the function assignSmartBucket
        /// ***********************************************************

        
        assignSmartBucket_iterative<T><<<numBlocks, threadsPerBlock, numUniqueBuckets * sizeof(T) +
                                     numUniqueBuckets * sizeof(double) + numUniqueBuckets * sizeof(unsigned int) +
                                      numBuckets * sizeof(unsigned int)>>>
                                      (d_vector, length, d_elementToBucket, d_slopes, d_pivotsLeft, d_pivotsRight,
                                       d_kthnumBuckets, d_bucketCount, numUniqueBuckets, numBuckets, offset);


        SAFEcuda("End of STEP 3\n");


        /// ***********************************************************
        /// **** STEP 4: IdentifyActiveBuckets
        /// **** Find the kth buckets
        /// **** and update their respective indices
        /// ***********************************************************

        sumCounts<<<numBuckets/threadsPerBlock, threadsPerBlock>>>(d_bucketCount, numBuckets, numBlocks);

        SAFEcuda("STEP 4, after sumCounts\n");

        findKBuckets(d_bucketCount, h_bucketCount, numBuckets, kVals, numKs, kthBucketScanner, kthBuckets, numBlocks);

        SAFEcuda("STEP 4, after findKBuckets");


        //display information
        printf("numKs: %d\n", numKs);
        //printf("h_bucketCount:\n");
        //for (int i = 0; i < numBuckets; i++)
          //  printf("%d, ", h_bucketCount[i]);
        //printf("\n");
        printf("kthBuckets: \n");
        for (int i = 0; i < numKs; i++)
            printf("%d, ", kthBuckets[i]);
        printf("\n");
        printf("kthBucketsScanner: \n");
        for (int i = 0; i < numKs; i++)
            printf("%d, ", kthBucketScanner[i]);
        printf("\n");
        printf("\n");





        updatekVals_iterative<T>(kVals, &numKs, output, kIndices, &length, &lengthOld, h_bucketCount, kthBuckets, kthBucketScanner,
                              reindexCounter, uniqueBuckets, uniqueBucketCounts, &numUniqueBuckets, &numUniqueBucketsOld,
                              tempKorderBucket, tempKorderIndeces, &tempKorderLength);

        SAFEcuda("STEP 4, after updatekVals\n");




        //display information
        printf("numKs: %d, length: %d, numUniqueBuckets: %d, tempKorderLength: %d\n", numKs, length, numUniqueBuckets, tempKorderLength);
        printf("numUniqueBucketsOld: %d\n", numUniqueBucketsOld);
        printf("uniqueBuckets:\n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", uniqueBuckets[i]);
        printf("\n");
        printf("uniqueBucketCounts:\n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", uniqueBucketCounts[i]);
        printf("\n");
        printf("reindexCounter:\n");
        for (int i = 0; i < numUniqueBuckets; i++)
            printf("%d, ", reindexCounter[i]);
        printf("\n");
        printf("\n");



        if (tempKorderLength > 0) {
            CUDA_CALL(hipMemcpy(d_tempKorderBucket, tempKorderBucket, tempKorderLength * sizeof(unsigned int),
                                 hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(d_tempKorderIndeces, tempKorderIndeces, tempKorderLength * sizeof(unsigned int),
                                 hipMemcpyHostToDevice));

            // potential to fix how many blocks to assign
            updateOutput_iterative<<<numBlocks, threadsPerBlock>>>(d_vector, d_elementToBucket, lengthOld, d_tempOutput,
                                                                   d_tempKorderBucket, tempKorderLength, offset);

            SAFEcuda("STEP 4, after updateOutput\n");

            CUDA_CALL(hipMemcpy(tempOutput, d_tempOutput, tempKorderLength * sizeof(T),
                                 hipMemcpyDeviceToHost));

            for (int i = 0; i < tempKorderLength; i++)
                output[tempKorderIndeces[i]] = tempOutput[i];
        }


        // display information
        printf("tempKorderIndeces: \n");
        for (int i = 0; i < tempKorderLength; i++)
            printf("%d, ", tempKorderIndeces[i]);
        printf("\n");
        printf("tempOutput: \n");
        for (int i = 0; i < tempKorderLength; i++)
            printf("%d, ", tempOutput[i]);
        printf("\n");
        printf("\n");

        printf("LeftKorderIndeces: \n");
        for (int i = 0; i < numKs; i++)
            printf("%d, ", kIndices[i]);
        printf("\n");
        printf("LeftKorder: \n");
        for (int i = 0; i < numKs; i++)
            printf("%d, ", kVals[i]);
        printf("\n");
        printf("\n");

        if (numKs <= 0)
            return 0;



        /// ***********************************************************
        /// **** STEP 5: Reduce
        /// **** Iteratively go through the loop to find correct
        /// **** order statistics and reduce the vector size
        /// ***********************************************************


        for (int j = 0; j < 4; j++) {


            printf("This is iteration %d\n", j+1);


            /// ***********************************************************
            /// **** STEP 5: Reduce
            /// **** Step 5.1: Copy active elements
            /// **** Copy the elements from the unique active buckets
            /// ***********************************************************

            CUDA_CALL(hipMemcpy(d_reindexCounter, reindexCounter,
                                 numUniqueBuckets * sizeof(unsigned int), hipMemcpyHostToDevice));
            CUDA_CALL(hipMemcpy(d_uniqueBuckets, uniqueBuckets,
                                 numUniqueBuckets * sizeof(unsigned int), hipMemcpyHostToDevice));

            reindexCounts<<<(int) ceil((float)numUniqueBuckets/threadsPerBlock), threadsPerBlock>>>
                            (d_bucketCount, numBuckets, numBlocks, d_reindexCounter, d_uniqueBuckets, numUniqueBuckets);

            SAFEcuda("STEP 5.1, after reindexCounts\n");

            copyElements_iterative<T><<<numBlocks, threadsPerBlock, numUniqueBuckets * sizeof(unsigned int)>>>
                                (d_vector, d_newvector, lengthOld, d_elementToBucket, d_uniqueBuckets, numUniqueBuckets,
                                 d_bucketCount, numBuckets, offset);

            SAFEcuda("STEP 5.1, after copyElements\n");

            swapPointers(&d_vector, &d_newvector);

            SAFEcuda("STEP 5.1, after swapPointers\n");


            //display information
            printf("numKs: %d, length: %d, numUniqueBuckets: %d, tempKorderLength: %d\n", numKs, length, numUniqueBuckets, tempKorderLength);
            printf("lengthOld: %d, numUniqueBucketsOld: %d\n", lengthOld, numUniqueBucketsOld);





            /// ***********************************************************
            /// **** STEP 5: Reduce
            /// **** Step 5.2: Update the pivots
            /// **** Update pivots to generate Pivots and Slopes in Step 5.3
            /// ***********************************************************

            CUDA_CALL(hipMemcpy(d_uniqueBuckets, uniqueBuckets, numUniqueBuckets * sizeof(unsigned int),
                                 hipMemcpyHostToDevice));


            // potential to fix how many blocks to assign
            updatePivots_iterative<T><<<numBlocks, threadsPerBlock>>>(d_pivotsLeft, d_newPivotsLeft, d_newPivotsRight,
                                                                   d_slopes, d_kthnumBuckets, d_uniqueBuckets,
                                                                   numUniqueBuckets, numUniqueBucketsOld, offset);

            SAFEcuda("STEP 5.2, after updatePivots\n");

            swapPointers(&d_pivotsLeft, &d_newPivotsLeft);
            swapPointers(&d_pivotsRight, &d_newPivotsRight);

            SAFEcuda("STEP 5.2, after swapPointers\n");


            /// ***********************************************************
            /// **** STEP 5: Reduce
            /// **** Step 5.3: create slopes and buckets offset
            /// **** create slopes and buckets offset for next iteration
            /// ***********************************************************

            CUDA_CALL(hipMemcpy(d_uniqueBucketCounts, uniqueBucketCounts, numUniqueBuckets * sizeof(unsigned int),
                                 hipMemcpyHostToDevice));


            // potential to fix how many blocks to assign
            generateBucketsandSlopes_iterative<<<numBlocks, threadsPerBlock>>>
                                                 (d_pivotsLeft, d_pivotsRight, d_slopes, d_uniqueBucketCounts,
                                                  numUniqueBuckets, d_kthnumBuckets, length, offset, numBuckets);

            SAFEcuda("STEP 5.3, after generateBucketandSlopes\n");

            CUDA_CALL(hipMemcpy(slopes, d_slopes, numUniqueBuckets * sizeof(double),
                                 hipMemcpyDeviceToHost));

            // make any slopes that were infinity due to division by zero (due to no
            //  difference between the two associated pivots) into zero, so all the
            //  values which use that slope are projected into a single bucket
            for (int i = 0; i < numUniqueBuckets; i++)
                if (isinf(slopes[i]))
                    slopes[i] = 0;

            CUDA_CALL(hipMemcpy(d_slopes, slopes, numUniqueBuckets * sizeof(double),
                                 hipMemcpyHostToDevice));



            //display information
            if (j < 4) {
                hipMemcpy(pivotsLeft, d_pivotsLeft, numUniqueBuckets * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(pivotsRight, d_pivotsRight, numUniqueBuckets * sizeof(int), hipMemcpyDeviceToHost);
                printf("PivotsLeft: \n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", pivotsLeft[i]);
                printf("\n");
                printf("PivotsRight: \n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", pivotsRight[i]);
                printf("\n");
                printf("\n");
            }


            //display information
            if (j < 4) {
                hipMemcpy(slopes, d_slopes, numUniqueBuckets * sizeof(double), hipMemcpyDeviceToHost);
                hipMemcpy(kthnumBuckets, d_kthnumBuckets, numUniqueBuckets * sizeof(unsigned int), hipMemcpyDeviceToHost);
                printf("slopes: \n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%lf, ", slopes[i]);
                printf("\n");
                printf("kthnumBuckets: \n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", kthnumBuckets[i]);
                printf("\n");
                printf("\n");
            }

            printf("length: %d, numUniqueBuckets: %d\n", length, numUniqueBuckets);

	    /*
            //display information
            if (j == 1) {
                int * h_vector = (int*)malloc(length * sizeof(int));
                hipMemcpy(h_vector, d_vector, length * sizeof(int), hipMemcpyDeviceToHost);
                printf("display vector\n");
                for (int i = 0; i < length; i++)
                    printf("%d, ", h_vector[i]);
                printf("\n");
                printf("\n");
            }
            */





            /// ***********************************************************
            /// **** STEP 5: Reduce
            /// **** Step 5.4: assign buckets
            /// **** assign elements to correct buckets in iteration
            /// ***********************************************************
            
             
            
           
            
            assignSmartBucket_iterative<T><<<numBlocks, threadsPerBlock, numUniqueBuckets * sizeof(T) +
                                                                         numUniqueBuckets * sizeof(double) +
                                                                         numUniqueBuckets * sizeof(unsigned int) +
                                                                         numBuckets * sizeof(unsigned int)>>>
                    (d_vector, length, d_elementToBucket, d_slopes, d_pivotsLeft, d_pivotsRight, d_kthnumBuckets,
                     d_bucketCount, numUniqueBuckets, numBuckets, offset);



            SAFEcuda("STEP 5.4, after assignSmartBucket\n");



            /*
            //display information
            //if (j == 0 || j == 1) {
            if (j == 1) {
                int * h_vector = (int*)malloc(length * sizeof(int));
                hipMemcpy(h_vector, d_vector, length * sizeof(int), hipMemcpyDeviceToHost);
                unsigned int * h_elementToBucket = (unsigned int*)malloc(length * sizeof(unsigned int));
                hipMemcpy(h_elementToBucket, d_elementToBucket, length * sizeof(unsigned int), hipMemcpyDeviceToHost);
                printf("display vector and element to bucket\n");
                for (int i = 0; i < length; i++)
                    printf("%d, %d;   ", h_vector[i], h_elementToBucket[i]);
                printf("\n");
                printf("\n");
            }
            */
             
            
            SAFEcuda("STEP 5.4, after prnt evrything\n"); 

            /*
            //display information
            if (j == 1) {
                unsigned int * h_elementToBucket = (unsigned int*)malloc(length * sizeof(unsigned int));
                hipMemcpy(h_elementToBucket, d_elementToBucket, length * sizeof(unsigned int), hipMemcpyDeviceToHost);
                printf("display elementToBucket\n");
                for (int i = 0; i < length; i++)
                    printf("%d, ", h_elementToBucket[i]);
                printf("\n");
                printf("\n");
            }
             */





            /// ***********************************************************
            /// **** STEP 5: Reduce
            /// **** Step 5.5: IdentifyActiveBuckets
            /// **** Find kth buckets and update their respective indices
            /// ***********************************************************

            sumCounts<<<numBuckets/threadsPerBlock, threadsPerBlock>>>(d_bucketCount, numBuckets, numBlocks);

            SAFEcuda("STEP 5.5, after sumCounts\n");

            findKBuckets(d_bucketCount, h_bucketCount, numBuckets, kVals, numKs, kthBucketScanner, kthBuckets, numBlocks);

            SAFEcuda("STEP 5.5, after findKBuckets\n");



            //display information
            if (j < 4) {
                printf("numKs: %d\n", numKs);
                //for (int i = 0; i < numBuckets; i++)
                  //  printf("%d, ", h_bucketCount[i]);
                //printf("\n");
                printf("kthBuckets: \n");
                for (int i = 0; i < numKs; i++)
                    printf("%d, ", kthBuckets[i]);
                printf("\n");
                printf("kthBucketsScanner: \n");
                for (int i = 0; i < numKs; i++)
                    printf("%d, ", kthBucketScanner[i]);
                printf("\n");
                printf("\n");
            }





            updatekVals_iterative<T>(kVals, &numKs, output, kIndices, &length, &lengthOld, h_bucketCount, kthBuckets, kthBucketScanner,
                                     reindexCounter, uniqueBuckets, uniqueBucketCounts, &numUniqueBuckets, &numUniqueBucketsOld,
                                     tempKorderBucket, tempKorderIndeces, &tempKorderLength);

            SAFEcuda("STEP 5.5, after updateKVals\n");




            //display information
            if (j < 4) {
                printf("numKs: %d, length: %d, numUniqueBuckets: %d, tempKorderLength: %d\n", numKs, length,
                       numUniqueBuckets, tempKorderLength);
                printf("uniqueBuckets:\n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", uniqueBuckets[i]);
                printf("\n");
                printf("uniqueBucketCounts:\n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", uniqueBucketCounts[i]);
                printf("\n");
                printf("reindexCounter:\n");
                for (int i = 0; i < numUniqueBuckets; i++)
                    printf("%d, ", reindexCounter[i]);
                printf("\n");
                printf("kVals:\n");
                for (int i = 0; i < numKs; i++)
                    printf("%d, ", kVals[i]);
                printf("\n");
                printf("\n");
            }



            if (tempKorderLength > 0) {
                CUDA_CALL(hipMemcpy(d_tempKorderBucket, tempKorderBucket, tempKorderLength * sizeof(unsigned int),
                                     hipMemcpyHostToDevice));
                CUDA_CALL(hipMemcpy(d_tempKorderIndeces, tempKorderIndeces, tempKorderLength * sizeof(unsigned int),
                                     hipMemcpyHostToDevice));

                // potential to fix how many blocks to assign
                updateOutput_iterative<<<numBlocks, threadsPerBlock>>>(d_vector, d_elementToBucket, lengthOld, d_tempOutput,
                                                                       d_tempKorderBucket, tempKorderLength, offset);

                SAFEcuda("STEP 5.5, after updateOutput\n");

                CUDA_CALL(hipMemcpy(tempOutput, d_tempOutput, tempKorderLength * sizeof(T),
                                     hipMemcpyDeviceToHost));

                for (int i = 0; i < tempKorderLength; i++)
                    output[tempKorderIndeces[i]] = tempOutput[i];

            }



            //display information
            if (j < 4) {
                printf("tempKorderIndeces: \n");
                for (int i = 0; i < tempKorderLength; i++)
                    printf("%d, ", tempKorderIndeces[i]);
                printf("\n");
                printf("tempOutput: \n");
                for (int i = 0; i < tempKorderLength; i++)
                    printf("%d, ", tempOutput[i]);
                printf("\n");
                printf("\n");
            }

            //display information
            if (j < 4) {
                printf("LeftKorderIndeces: \n");
                for (int i = 0; i < numKs; i++)
                    printf("%d, ", kIndices[i]);
                printf("\n");
                printf("LeftKorder: \n");
                for (int i = 0; i < numKs; i++)
                    printf("%d, ", kVals[i]);
                printf("\n");
                printf("\n");
            }



            if (numKs <= 0)
                return 0;

            // display information
            printf("Done iteration %d\n\n", j+1);

        }


        CUDA_CALL(hipMemcpy(d_kthBuckets, kthBuckets, numKs * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

        updateOutput_iterative<<<numBlocks, threadsPerBlock>>>(d_vector, d_elementToBucket, lengthOld, d_tempOutput,
                                                               d_kthBuckets, numKs, offset);

        SAFEcuda("Exit Iteration, after updateOutput\n");

        CUDA_CALL(hipMemcpy(tempOutput, d_tempOutput, numKs * sizeof(T),
                             hipMemcpyDeviceToHost));

        for (int i = 0; i < numKs; i++)
            output[kIndices[i]] = tempOutput[i];


        /*
        int * h_vector = (int *)malloc(length * sizeof(int));
        hipMemcpy(h_vector, d_vector, length * sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 0; i < length; i++) {
            printf("%d   ", h_vector[i]);
        }

        printf("\n");
         */

        free(slopes);
        free(pivots);
        free(pivotsLeft);
        free(pivotsRight);
        free(newPivotsLeft);
        free(newPivotsRight);
        free(h_bucketCount);
        free(kthBuckets);
        free(kthBucketScanner);
        free(kIndices);
        free(uniqueBuckets);
        free(uniqueBucketCounts);
        free(reindexCounter);
        free(kthnumBuckets);
        free(tempOutput);
        free(tempKorderBucket);
        free(tempKorderIndeces);


        hipFree(d_slopes);
        hipFree(d_pivots);
        hipFree(d_pivotsLeft);
        hipFree(d_pivotsRight);
        hipFree(d_newPivotsLeft);
        hipFree(d_newPivotsRight);
        hipFree(d_elementToBucket);
        hipFree(d_bucketCount);
        hipFree(d_newvector);
        hipFree(d_kVals);
        hipFree(d_kthBuckets);
        hipFree(d_kthBucketScanner);
        hipFree(d_kIndices);
        hipFree(d_uniqueBuckets);
        hipFree(d_uniqueBucketCounts);
        hipFree(d_reindexCounter);
        hipFree(d_kthnumBuckets);
        hipFree(d_tempOutput);
        hipFree(d_tempKorderBucket);
        hipFree(d_tempKorderIndeces);

        SAFEcuda("Exit Iteration, after free\n");


        return 0;
    }

}


int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}


int main() {

    // test for iterativeSMOS
    int threadsPerBlock = 1024;
    int numBlocks = 12; // figure out how to query the system and dp.XXX using hipGetDeviceProperties
    int numTotalBuckets = 8192;
//    int offset = threadsPerBlock * numBlocks;
//    int numBuckets = 8192;

    int length = 1000;
    float * h_vector = (float*)malloc(length * sizeof(float));
    for (int i = 0; i < length; i++) {
        h_vector[i] = (float)(rand()/(float)RAND_MAX);
        printf("\nh_vec[%d]=%f",i,h_vector[i]);
    }
    
    qsort(h_vector, length, sizeof(float), cmpfunc);
    
    for (int i = 0; i < length; i++) {
        printf("\nh_vec[%d]=%f",i,h_vector[i]);
    }
    
/*    unsigned int * h_vector = (unsigned int*)malloc(length * sizeof(unsigned int));
    for (int i = 0; i < length; i++) {
        h_vector[i] = i + 1;
    }

    unsigned int * d_vector;
    hipMalloc(&d_vector, length * sizeof(unsigned int));
    hipMemcpy(d_vector, h_vector, length * sizeof(unsigned int), hipMemcpyHostToDevice);
    */
    float * d_vector;
    hipMalloc(&d_vector, length * sizeof(float));
    hipMemcpy(d_vector, h_vector, length * sizeof(float), hipMemcpyHostToDevice);
    for (int ksize=11; ksize<399; ksize+=9) { 
         printf("\n \n kVals:\n");
         unsigned int numKs = (unsigned int) ksize;
         unsigned int * kVals = (unsigned int *)malloc(numKs * sizeof(unsigned int));
         unsigned int * oldkVals = (unsigned int *)malloc(numKs * sizeof(unsigned int));
         unsigned int notready;
         
         
         printf("\n\n\n\n\n\n ############################################################################# \n\n New prblm with numKs = %d.\n\n", numKs);
         
         
         for (int i = 0; i < numKs; i++) {
           notready = 1;
           while (notready) {
             kVals[i] = rand() % length; //i*2+1; // * 100 + 1;
             notready = 0;
             if (i>0) {
               for (int ii=0; ii<i; ii++){
                 notready += (kVals[ii]==kVals[i]);
               }  // ends for ii
             } // ends if i>0
           } // ends while notready
           printf("%d  ", kVals[i]);
           oldkVals[i]=kVals[i];
         }

         float * output = (float*)malloc(numKs * sizeof(float));
//         unsigned int * output = (unsigned int*)malloc(numKs * sizeof(unsigned int));

         for (int i = 0; i < numKs; i++) {
             output[i] = 0;
         }

         IterativeSMOS::iterativeSMOS(d_vector, length ,kVals, numKs, output, numBlocks, threadsPerBlock, numTotalBuckets, 17);

         for (int i = 0; i < numKs; i++) {
//             std::cout << output[i] ;
//             printf("%d   ", output[i]);
             printf("%f   ", output[i]);
         }
         printf("\n");
         int numwrong = 0;
         for (int i = 0; i < numKs; i++) {
           printf("\nkVals[%d]=%d,  ", i, oldkVals[i]);
           if (h_vector[oldkVals[i]-1] != output[i])
             printf("\n WRONG For i=%d, hvec=%f and out=%f  ",i, h_vector[oldkVals[i]-1],output[i]);
             numwrong++;
         }
         print("It got %d wrong.", numwrong);
         free(kVals);
         free(oldkVals);
         free(output);
    }
    
    free(h_vector);
    hipFree(d_vector);



    return 0;
}
