/* Based on timingFunctions.cu */

template <typename T>
 struct results_t{
  float time;
  T * vals;
};

template<typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T* h_vec, uint size, results_t<T> **result){
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *)malloc(sizeof(results_t<T>));
}

template<typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, T* d_vec, results_t<T> *result, float time){
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->time = time;
  //   hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////
//          THE SORT AND CHOOSE TIMING FUNCTION
/////////////////////////////////////////////////////////////////
template<typename T>
results_t<T>* timeSortAndChooseMultiselect(T *h_vec, uint numElements, uint * kVals, uint kCount){


  T* d_vec;
  T returnValueFromSelect;
  results_t<T> *result;
  float time;
  hipEvent_t start, stop;
 
  setupForTiming(start,stop, &d_vec, h_vec, numElements, &result);

  thrust::device_ptr<T> dev_ptr(d_vec);
  hipEventRecord(start, 0);

  thrust::sort(dev_ptr, dev_ptr + numElements);

  hipMemcpy(h_vec, d_vec, numElements * sizeof(T), hipMemcpyDeviceToHost);
  for (int i = 0; i < kCount; i++)
    result->vals[i] = h_vec[numElements - kVals[i]]; 

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start,stop);


  wrapupForTiming(start,stop, d_vec, result, time);
  return result;
}

// FUNCTION TO TIME BUCKET MULTISELECT
template<typename T>
results_t<T>* timeBucketMultiselect (T *h_vec, uint numElements, uint * kVals, uint kCount){


  T* d_vec;
  T returnValueFromSelect;
  results_t<T> *result;
  float time;
  hipEvent_t start, stop;
  hipDeviceProp_t dp;

  hipGetDeviceProperties(&dp,0);


  setupForTiming(start,stop, &d_vec, hostVec, size, &result);

  hipEventRecord(start, 0);

  BucketSelect::bucketMultiselectWrapper(d_vec, numElements, kVals, kCount, result->vals, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);


  wrapupForTiming(start,stop, deviceVec, result, time);
  return result;
}
