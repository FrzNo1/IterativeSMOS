/* Based on timingFunctions.cu */
#include <stdlib.h>

template <typename T>
 struct results_t {
  float time;
  T * vals;
};

template<typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T* h_vec, uint size, results_t<T> **result, uint kCount) {
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *) malloc(sizeof(results_t<T>));
}

template<typename T>
void setupForTimingSortAndChoose(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T* h_vec, uint size, results_t<T> **result, uint kCount) {
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *) malloc(sizeof(results_t<T>));
  (*result)->vals = (T *) malloc (kCount * sizeof (T));
}

template<typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, T* d_vec, results_t<T> *result, float time, T * value) {
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->time = time;
  result->vals = value;
  //   hipDeviceSynchronize();
}

template<typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, T* d_vec, results_t<T> *result, float time) {
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->time = time;
  //   hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////
//          THE SORT AND CHOOSE TIMING FUNCTION
/////////////////////////////////////////////////////////////////
template<typename T>
results_t<T>* timeSortAndChooseMultiselect(T *h_vec, uint numElements, uint * kVals, uint kCount) {

  T* d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;
 
  setupForTimingSortAndChoose(start, stop, &d_vec, h_vec, numElements, &result, kCount);

  thrust::device_ptr<T> dev_ptr(d_vec);
  hipEventRecord(start, 0);

  thrust::sort(dev_ptr, dev_ptr + numElements);

  hipMemcpy(h_vec, d_vec, numElements * sizeof(T), hipMemcpyDeviceToHost);
  for (int i = 0; i < kCount; i++)
    result->vals[i] = h_vec[numElements - kVals[i]];

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);


  wrapupForTiming(start, stop, d_vec, result, time);
  return result;
}

// FUNCTION TO TIME BUCKET MULTISELECT
template<typename T>
results_t<T>* timeBucketMultiselect (T * h_vec, uint numElements, uint * kVals, uint kCount) {


  T* d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;
  hipDeviceProp_t dp;
  T* result_vals = (T *) malloc (kCount * sizeof (T));

  hipGetDeviceProperties(&dp,0);


  setupForTiming(start, stop, &d_vec, h_vec, numElements, &result, kCount);

  hipEventRecord(start, 0);

  printf("start here\n");
  // void bucketMultiselectWrapper (T * d_vector, int length, uint * kVals_ori, uint kCount, T * outputs, int blocks, int threads) { 
  BucketMultiselect::bucketMultiselectWrapper(d_vec, numElements, kVals, kCount, result_vals, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  printf("start here\n");
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);


  wrapupForTiming(start, stop, d_vec, result, time, result_vals);
  return result;
}
