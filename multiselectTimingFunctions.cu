#include "hip/hip_runtime.h"
/* Based on timingFunctions.cu */
#include <stdlib.h>

#define MAX_THREADS_PER_BLOCK 1024

#define CUDA_CALL(x) do { if((x) != hipSuccess) {    \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

template <typename T>
 struct results_t {
  float time;
  T * vals;
};

template <typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop, T * h_vec, T ** d_vec, results_t<T> ** result, uint numElements, uint kCount) {
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMalloc(d_vec, numElements * sizeof(T));
  hipMemcpy(*d_vec, h_vec, numElements * sizeof(T), hipMemcpyHostToDevice);

  *result = (results_t<T> *) malloc (sizeof (results_t<T>));
  (*result)->vals = (T *) malloc (kCount * sizeof (T));
}

template <typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop, float time, results_t<T> * result) {
  result->time = time;
  hipEventDestroy(start);
  hipEventDestroy(stop);
  //   hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////
//          THE SORT AND CHOOSE TIMING FUNCTION
/////////////////////////////////////////////////////////////////


template <typename T>
__global__ void copyInChunk(T * outputVector, T * inputVector, uint * kList, uint kListCount, uint numElements) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < kListCount) 
    outputVector[idx] = inputVector[numElements - kList[idx]];
  
}

template<typename T>
results_t<T>* timeSortAndChooseMultiselect(T * h_vec, uint numElements, uint * kVals, uint kCount) {
  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);

  hipEventRecord(start, 0);
  thrust::device_ptr<T> dev_ptr(d_vec);
  thrust::sort(dev_ptr, dev_ptr + numElements);

  /*
  for (int i = 0; i < kCount; i++)
    hipMemcpy(result->vals + i, d_vec + (numElements - kVals[i]), sizeof (T), hipMemcpyDeviceToHost);
  */
  
  T * d_output;
  uint * d_kList;

  hipMalloc (&d_output, kCount * sizeof (T));
  hipMalloc (&d_kList, kCount * sizeof(uint));
  hipMemcpy (d_kList, kVals, kCount * sizeof (uint), hipMemcpyHostToDevice);

  int threads = MAX_THREADS_PER_BLOCK;
  if (kCount < threads)
    threads = kCount;
  int blocks = (int) ceil (kCount / (float) threads);

  copyInChunk<T><<<blocks, threads>>>(d_output, d_vec, d_kList, kCount, numElements);
  hipMemcpy (result->vals, d_output, kCount * sizeof (T), hipMemcpyDeviceToHost);
//••••••••••••••••••••••
  //printf("first result: %u \n", result->vals);

  hipFree(d_output);
  hipFree(d_kList); 
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}


// FUNCTION TO TIME BUCKET MULTISELECT
template<typename T>
results_t<T>* timeBucketMultiselect (T * h_vec, uint numElements, uint * kVals, uint kCount) {
  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp, 0);

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);
 
  hipEventRecord(start, 0);

  // bucketMultiselectWrapper (T * d_vector, int length, uint * kVals_ori, uint kCount, T * outputs, int blocks, int threads)
  BucketMultiselect::bucketMultiselectWrapper(d_vec, numElements, kVals, kCount, result->vals, dp.multiProcessorCount, dp.maxThreadsPerBlock);

  printf("BMS");
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}


// FUNCTION TO TIME NAIVE BUCKET MULTISELECT
template<typename T>
results_t<T>* timeNaiveBucketMultiselect (T * h_vec, uint numElements, uint * kVals, uint kCount) {
  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);

  hipEventRecord(start, 0);
  thrust::device_ptr<T> dev_ptr(d_vec);
  thrust::sort(dev_ptr, dev_ptr + numElements);

  for (int i = 0; i < kCount; i++)
    hipMemcpy(result->vals + i, d_vec + (numElements - kVals[i]), sizeof (T), hipMemcpyDeviceToHost);
   

  printf("hello");
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}

/***************************************
********* TOP K SELECT
****************************************/

template<typename T>
results_t<T>* timeSortAndChooseTopkselect(T * h_vec, uint numElements, uint kCount) {
  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);

  hipEventRecord(start, 0);
  thrust::device_ptr<T> dev_ptr(d_vec);
  thrust::sort(dev_ptr, dev_ptr + numElements, thrust::greater<T>());

  hipMemcpy(result->vals, d_vec, kCount * sizeof(T), hipMemcpyDeviceToHost);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}

// FUNCTION TO TIME RANDOMIZED TOP K SELECT
template<typename T>
results_t<T>* timeRandomizedTopkselect (T * h_vec, uint numElements, uint kCount) {
  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp, 0);

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);
 
  hipEventRecord(start, 0);
  result->vals = randomizedTopkSelectWrapper(d_vec, numElements, kCount);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}

// FUNCTION TO TIME BUCKET TOP K SELECT
template<typename T>
results_t<T>* timeBucketTopkselect (T * h_vec, uint numElements, uint kCount) {
  // initialize ks
  uint * kVals = (uint *) malloc(kCount*sizeof(T));
  for (uint i = 0; i < kCount; i++)
    kVals[i] = i+1;

  T * d_vec;
  results_t<T> * result;
  float time;
  hipEvent_t start, stop;
  hipDeviceProp_t dp;
  hipGetDeviceProperties(&dp, 0);

  setupForTiming(start, stop, h_vec, &d_vec, &result, numElements, kCount);
 
  hipEventRecord(start, 0);

  BucketMultiselect::bucketMultiselectWrapper(d_vec, numElements, kVals, kCount, result->vals, dp.multiProcessorCount, dp.maxThreadsPerBlock);
 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  wrapupForTiming(start, stop, time, result);
  hipFree(d_vec);
  return result;
}
