#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>

void createRandomMatrix(float * d_A, int size, int seed) {
  hiprandGenerator_t gen;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateUniform(gen, d_A, size);

  hiprandDestroyGenerator(gen);
}

template <typename T>
__global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
  *(in + threadIdx.x) = *(list + ((int) (*(in + threadIdx.x) * size)));
}

void printStuff (float * d_list, int size) {
  float * p = (float *) malloc (sizeof (float) * size);

  hipMemcpy (p, d_list, sizeof (float) * size, hipMemcpyDeviceToHost);

  for (int i = 0; i < 20; i++)
    printf("%f\n", *(p+i));
  
  free(p);
}

  template <typename T>
  void generatePivots (T * pivots, double * slopes, T * d_list, int numElements, int numPivots, int sampleSize, T min, T max) {

    T * d_randoms;
    int pivotOffset = (sampleSize / (numPivots - 1));

    hipMalloc ((void **) &d_randoms, sizeof (T) * sampleSize);
  
    createRandomMatrix (d_randoms, sampleSize, 1);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<1, sampleSize>>>(d_randoms, d_list, numElements);

    pivots[0] = min; 
    pivots[numPivots-1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randoms);
    thrust::sort(randoms_ptr, randoms_ptr + sampleSize);

    hipDeviceSynchronize();

    for (int i = 1; i < numPivots - 1; i++) {
      hipMemcpy (pivots + i, d_randoms + pivotOffset * i, sizeof (T), hipMemcpyDeviceToHost);
      slopes[i-1] = pivotOffset /(pivots[i] - pivots[i-1]);
    }
    
    slopes[numPivots-2] = pivotOffset / (pivots[numPivots-1] - pivots[numPivots-2]);
  
    hipFree(d_randoms);
  }



int main() {

  for (int i = 0; i < 1; i++) {
    int numElements = 1000000;
    int sampleSize = 1024;
    int numSplitters = 9;
  
    float splitters[numSplitters];
    double slopes[numSplitters - 1];
  
    float * list = (float *) malloc (numElements * sizeof (float));

    // initialize array
    for (int j = 0; j < numElements; j++)
      list[j] = (float) j;

    float * d_list;
    hipMalloc ((void **) &d_list, numElements * sizeof (float));
    hipMemcpy(d_list, list, numElements * sizeof (float), hipMemcpyHostToDevice);

    hipEvent_t start3, stop3;
    float time3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3,0);

    /*    template <typename T>
          void generatePivots (T * pivots, double * slopes, T * d_list, int numElements, int numPivots, int sampleSize, T min, T max) */
       
    generatePivots<float>(splitters, slopes, d_list, numElements, numSplitters, sampleSize, list[0], list[numElements-1]);

    hipDeviceSynchronize();
    hipEventRecord(stop3,0);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&time3, start3, stop3);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);
    
    free(list);
    hipFree(d_list);
   
    printf("time = %f\n", time3);
  }

  
  return 0;
}
