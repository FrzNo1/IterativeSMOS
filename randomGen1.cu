#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <stdio.h>
#include <sys/time.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>

#define FLOAT 1
#define DOUBLE 1
#define INT 1

///**** MonteCarlo.cu ****
// we could vary M sdf& N to find the perf sweet spot

__host__ __device__
unsigned int hash(unsigned int a)
{
  a = (a+0x7ed55d16) + (a<<12);
  a = (a^0xc761c23c) ^ (a>>19);
  a = (a+0x165667b1) + (a<<5);
  a = (a+0xd3a2646c) ^ (a<<9);
  a = (a+0xfd7046c5) + (a<<3);
  a = (a^0xb55a4f09) ^ (a>>16);
  return a;
}


/*
struct RandomNumberFunctor :
  public thrust::binary_function<unsigned int, int, double>
{
  unsigned int mainSeed;
  unsigned int sizeOfVector;

  RandomNumberFunctor(unsigned int _mainSeed, int _sizeOfVector) :
    mainSeed(_mainSeed), sizeOfVector(_sizeOfVector){}
  
  __host__ __device__
  float operator()(unsigned int threadIdx)
  {
    unsigned int seed = hash(threadIdx) * mainSeed;

    thrust::default_random_engine rng(seed);
    rng.discard(threadIdx);
    thrust::uniform_real_distribution<double> u(0, sizeOfVector);

    return u(rng);
  }
};
*/

struct RandomNumberFunctor :
  public thrust::unary_function<unsigned int, float>
{
  unsigned int mainSeed;

  RandomNumberFunctor(unsigned int _mainSeed) :
    mainSeed(_mainSeed) {}
  
  __host__ __device__
  float operator()(unsigned int threadIdx)
  {
    unsigned int seed = hash(threadIdx) * mainSeed;

    thrust::default_random_engine rng(seed);
    rng.discard(threadIdx);
    thrust::uniform_real_distribution<float> u(0,1);

    return u(rng);
  }
};


///**** MonteCarlo.cu ****
template <typename T>
void createRandomVector(T * d_vec, int size) {
  timeval t1;
  uint seed;

  gettimeofday(&t1, NULL);
  seed = t1.tv_usec * t1.tv_sec;
  
  thrust::device_ptr<T> d_ptr(d_vec);
  thrust::transform(thrust::counting_iterator<uint>(0),thrust::counting_iterator<uint>(size),
                    d_ptr, RandomNumberFunctor(seed));
}

template <typename T>
void createRandomVectorCurand(T * d_A, int size) {
  hiprandGenerator_t gen;
  timeval t1;
  uint seed;

  gettimeofday(&t1, NULL);
  seed = t1.tv_usec * t1.tv_sec;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hiprandGenerateUniform(gen, d_A, size);

  hiprandDestroyGenerator(gen);
}

template <typename T>
__global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
  *(in + blockIdx.x*blockDim.x + threadIdx.x) = *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
}


__global__ void enlargeIndexAndGetElements (float * in, uint * out, uint * list, int size) {
  *(out + blockIdx.x * blockDim.x + threadIdx.x) = (uint) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
}


template <typename T>
void printStuff (T * d_list, int size) {
  T * p = (T *) malloc (sizeof (T) * size);

  hipMemcpy (p, d_list, sizeof(T)* size, hipMemcpyDeviceToHost);

  for (int i = 0; i < 20; i++)
    // printf("%lf\n", *(p+i));
    std::cout << *(p+i) << "\n";
  
  free(p);
}
/*

  template <typename T>
  void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector, int numPivots, int sizeOfSample, T min, T max) {

  int maxThreads = 1024;

  double * d_randoms;
  hipMalloc ((void **) &d_randoms, sizeof (double) * sizeOfSample);

  int numSmallBuckets = (sizeOfSample / (numPivots - 1));
  
  createRandomVectorThrust(d_randoms, sizeOfSample, sizeOfVector);

  // converts randoms floats into elements from necessary indices
  getElements<<<(sizeOfSample/maxThreads), maxThreads>>>(d_randoms, d_list, sizeOfVector);

  pivots[0] = (T) min; 
  pivots[numPivots-1] = (T) max;

  thrust::device_ptr<double>randoms_ptr(d_randoms);
  thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

  hipDeviceSynchronize();

  double holder;

  for (int i = 1; i < numPivots - 1; i++) {    
  hipMemcpy (&holder, (d_randoms + numSmallBuckets * i), sizeof (double), hipMemcpyDeviceToHost);     
  *(pivots + i) = (T) holder;
  slopes[i-1] = numSmallBuckets /(pivots[i] - pivots[i-1]);
  }
    
  slopes[numPivots-2] = numSmallBuckets / (pivots[numPivots-1] - pivots[numPivots-2]);

  hipFree(d_randoms);
  }
*/
template <typename T>
void generatePivots (uint * pivots, double * slopes, uint * d_list, int sizeOfVector, int numPivots, int sizeOfSample, uint min, uint max) {
  
  int maxThreads = 1024;
  float * d_randomFloats;
  uint * d_randomInts;
  
  int pivotOffset = (sizeOfSample / (numPivots - 1));

  hipMalloc ((void **) &d_randomFloats, sizeof (float) * sizeOfSample);
  
  d_randomInts = (uint *) d_randomFloats;

  createRandomVector (d_randomFloats, sizeOfSample);

  // converts randoms floats into elements from necessary indices
  enlargeIndexAndGetElements<<<(sizeOfSample/maxThreads), maxThreads>>>(d_randomFloats, d_randomInts, d_list, sizeOfVector);


  pivots[0] = min;
  pivots[numPivots-1] = max;

  thrust::device_ptr<T>randoms_ptr(d_randomInts);
  thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

  hipDeviceSynchronize();

  for (int i = 1; i < numPivots - 1; i++) {
    hipMemcpy (pivots + i, d_randomInts + pivotOffset * i, sizeof (uint), hipMemcpyDeviceToHost);
    slopes[i-1] = pivotOffset /(pivots[i] - pivots[i-1]);
  }
    
  slopes[numPivots-2] = pivotOffset / (pivots[numPivots-1] - pivots[numPivots-2]);
  
  hipFree(d_randomInts);
}

template <typename T>
void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector, int numPivots, int sizeOfSample, T min, T max) {

  int maxThreads = 1024;
  T * d_randoms;
  int pivotOffset = (sizeOfSample / (numPivots - 1));

  hipMalloc ((void **) &d_randoms, sizeof (T) * sizeOfSample);
  
  createRandomVector (d_randoms, sizeOfSample);

  // converts randoms floats into elements from necessary indices
  enlargeIndexAndGetElements<<<(sizeOfSample/maxThreads), maxThreads>>>(d_randoms, d_list, sizeOfVector);

  pivots[0] = min;
  pivots[numPivots-1] = max;

  thrust::device_ptr<T>randoms_ptr(d_randoms);
  thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

  hipDeviceSynchronize();

  // TODO:  check to make sure pivots aren't repeated, since that crashes our program currently
  //  note:  need to implement everything in both generatePivots functions

  // set pivots next to the min and max pivots
  hipMemcpy (pivots + 1, d_randoms + 1, sizeof (T), hipMemcpyDeviceToHost);
  hipMemcpy (pivots + numPivots - 2, d_randoms + sizeOfSample - 2, sizeof (T), hipMemcpyDeviceToHost);
  slopes[0] = pivotOffset / (double) (pivots[1] - pivots[0]);

  for (int i = 2; i < numPivots - 2; i++) {
    hipMemcpy (pivots + i, d_randoms + pivotOffset * i, sizeof (T), hipMemcpyDeviceToHost);

  }




  for (int i = 1; i < numPivots - 1; i++) {

    slopes[i-1] = pivotOffset / (pivots[i] - pivots[i-1]);
  }

  slopes[numPivots-3] = pivotOffset / (double) (pivots[numPivots-2] - pivots[numPivots-3]);
  slopes[numPivots-2] = pivotOffset / (pivots[numPivots-1] - pivots[numPivots-2]);
  
  hipFree(d_randoms);
}



/***** GENERALIZED VERSION *****/

/*
template <typename T>
__global__ void getElements (double * in, T * list, int size) {
  // *(in + blockIdx.x*numThreads + threadIdx.x) = *(list + ((int) (*(in + blockIdx.x*numThreads + threadIdx.x) * size)));
  // printf("%lf\n", *(in + blockIdx.x*blockDim.x + threadIdx.x));

  *(in + blockIdx.x*blockDim.x + threadIdx.x) = (double) *(list + (int) *(in + blockIdx.x*blockDim.x + threadIdx.x));
  // printf("%lf\n", *(in + blockIdx.x*blockDim.x + threadIdx.x));
}

template <typename T>
void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector, int numPivots, int sizeOfSample, T min, T max) {

  int maxThreads = 1024;

  double * d_randoms;
  hipMalloc ((void **) &d_randoms, sizeof (double) * sizeOfSample);

  int numSmallBuckets = (sizeOfSample / (numPivots - 1));
  
  createRandomVectorThrust(d_randoms, sizeOfSample, sizeOfVector);

  // converts randoms floats into elements from necessary indices
  getElements<<<(sizeOfSample/maxThreads), maxThreads>>>(d_randoms, d_list, sizeOfVector);

  pivots[0] = (T) min;
  pivots[numPivots-1] = (T) max;

  thrust::device_ptr<double>randoms_ptr(d_randoms);
  thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

  hipDeviceSynchronize();

  double holder;

  for (int i = 1; i < numPivots - 1; i++) {
    hipMemcpy (&holder, (d_randoms + numSmallBuckets * i), sizeof (double), hipMemcpyDeviceToHost);
    *(pivots + i) = (T) holder;
    slopes[i-1] = numSmallBuckets /(pivots[i] - pivots[i-1]);
  }
    
  slopes[numPivots-2] = numSmallBuckets / (pivots[numPivots-1] - pivots[numPivots-2]);

  hipFree(d_randoms);
}
*/


int main() {

  for (int i = 0; i < 1; i++) {
    int sizeOfVector = 100000000;
    int sizeOfSample = 1024;
    int numPivots = 17;
    
    //********* TEST FLOAT **********//
    if (FLOAT) {
      float floatPivots[numPivots];
      double floatSlopes[numPivots - 1];
  
      float * floatList = (float *) malloc (sizeOfVector * sizeof (float));

      // initialize array
      for (int j = 0; j < sizeOfVector; j++)
        floatList[j] =  (float) j;

      float * d_floatList;
      hipMalloc ((void **) &d_floatList, sizeOfVector * sizeof (float));
      hipMemcpy(d_floatList, floatList, sizeOfVector * sizeof (float), hipMemcpyHostToDevice);

      hipEvent_t start3, stop3;
      float time3;
      hipEventCreate(&start3);
      hipEventCreate(&stop3);
      hipEventRecord(start3,0);

      generatePivots<float>(floatPivots, floatSlopes, d_floatList, sizeOfVector, numPivots, sizeOfSample, floatList[0], floatList[sizeOfVector-1]);

      hipDeviceSynchronize();
      hipEventRecord(stop3,0);
      hipEventSynchronize(stop3);
      hipEventElapsedTime(&time3, start3, stop3);
      hipEventDestroy(start3);
      hipEventDestroy(stop3);
    
      free(floatList);
      hipFree(d_floatList);

      printf("\n\nfloat pivots:\n");
      for (int i = 0; i < numPivots; i++) 
        //std::cout << floatPivots[i] << '\n';
        printf("%f\n", floatPivots[i]);

      printf("\nfloat time = %f\n\n", time3);
    }

    //********* TEST DOUBLE **********//
    if (DOUBLE) {
      double doublePivots[numPivots];
      double doubleSlopes[numPivots - 1];

      double * doubleList = (double *) malloc (sizeOfVector * sizeof (double));

      // initialize array
      for (int j = 0; j < sizeOfVector; j++)
        doubleList[j] =  (double) j;

      double * d_doubleList;
      hipMalloc ((void **) &d_doubleList, sizeOfVector * sizeof (double));
      hipMemcpy(d_doubleList, doubleList, sizeOfVector * sizeof (double), hipMemcpyHostToDevice);

      hipEvent_t start1, stop1;
      float time1;
      hipEventCreate(&start1);
      hipEventCreate(&stop1);
      hipEventRecord(start1,0);

      generatePivots<double>(doublePivots, doubleSlopes, d_doubleList, sizeOfVector, numPivots, sizeOfSample, doubleList[0], doubleList[sizeOfVector-1]);

      hipDeviceSynchronize();
      hipEventRecord(stop1,0);
      hipEventSynchronize(stop1);
      hipEventElapsedTime(&time1, start1, stop1);
      hipEventDestroy(start1);
      hipEventDestroy(stop1);
    
      free(doubleList);
      hipFree(d_doubleList);

      printf("\n\ndouble pivots:\n");
      for (int i = 0; i < numPivots; i++) 
        //std::cout << floatPivots[i] << '\n';
        printf("%lf\n", doublePivots[i]);

      printf("\ndouble time = %f\n\n", time1);

    }

    if (INT) {
      uint intPivots[numPivots];
      double intSlopes[numPivots - 1];

      uint * intList = (uint *) malloc (sizeOfVector * sizeof (uint));

      // initialize array
      for (int j = 0; j < sizeOfVector; j++)
        intList[j] =  (uint) j;

      uint * d_intList;
      hipMalloc ((void **) &d_intList, sizeOfVector * sizeof (uint));
      hipMemcpy(d_intList, intList, sizeOfVector * sizeof (uint), hipMemcpyHostToDevice);

      hipEvent_t start2, stop2;
      float time2;
      hipEventCreate(&start2);
      hipEventCreate(&stop2);
      hipEventRecord(start2,0);

      generatePivots<uint>(intPivots, intSlopes, d_intList, sizeOfVector, numPivots, sizeOfSample, intList[0], intList[sizeOfVector-1]);

      hipDeviceSynchronize();
      hipEventRecord(stop2,0);
      hipEventSynchronize(stop2);
      hipEventElapsedTime(&time2, start2, stop2);
      hipEventDestroy(start2);
      hipEventDestroy(stop2);
    
      free(intList);
      hipFree(d_intList);

      printf("\n\nint pivots:\n");
      for (int i = 0; i < numPivots; i++) 
        //std::cout << intPivots[i] << '\n';
        printf("%u\n", intPivots[i]);
      
      printf("\nint time = %f\n\n", time2);
      
    }

  }
  return 0;
}
